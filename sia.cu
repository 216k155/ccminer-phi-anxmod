#include "hip/hip_runtime.h"
/**
 * Blake2-B CUDA Implementation
 *
 * tpruvot@github July 2016
 *
 */

#include <miner.h>

#include <string.h>
#include <stdint.h>

#include <sph/blake2b.h>

#include <cuda_helper.h>
#include <cuda_vectors.h>

#define TPB 512
#define NBN 2

static uint32_t *d_resNonces[MAX_GPUS];
static uint32_t *h_resNonces[MAX_GPUS];

static __constant__ uint2 _ALIGN(16) c_data[10];
static __constant__ uint2 _ALIGN(16) c_v[16];

static __constant__ const uint32_t blake2b_sigma[12][16] = {
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } , { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  } ,
	{ 11, 8,  12, 0,  5,  2,  15, 13, 10, 14, 3,  6,  7,  1,  9,  4  } , { 7,  9,  3,  1,  13, 12, 11, 14, 2,  6,  5,  10, 4,  0,  15, 8  } ,
	{ 9,  0,  5,  7,  2,  4,  10, 15, 14, 1,  11, 12, 6,  8,  3,  13 } , { 2,  12, 6,  10, 0,  11, 8,  3,  4,  13, 7,  5,  15, 14, 1,  9  } ,
	{ 12, 5,  1,  15, 14, 13, 4,  10, 0,  7,  6,  3,  9,  2,  8,  11 } , { 13, 11, 7,  14, 12, 1,  3,  9,  5,  0,  15, 4,  8,  6,  2,  10 } ,
	{ 6,  15, 14, 9,  11, 3,  0,  8,  12, 2,  13, 7,  1,  4,  10, 5  } , { 10, 2,  8,  4,  7,  6,  1,  5,  15, 11, 9,  14, 3,  12, 13, 0  } ,
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } , { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  }
};

extern "C" void blake2b_hash(void *output, const void *input)
{
	uint8_t _ALIGN(64) hash[32];
	blake2b_ctx ctx;

	blake2b_init(&ctx, 32, NULL, 0);
	blake2b_update(&ctx, input, 80);
	blake2b_final(&ctx, hash);

	memcpy(output, hash, 32);
}

// ----------------------------------------------------------------

__device__ __forceinline__
static void G(const int r, const int i, uint2 &a, uint2 &b, uint2 &c, uint2 &d,const uint2 m[16])
{
	a = a + b + m[ blake2b_sigma[r][2*i] ];
	d = SWAPUINT2( d ^ a );
	c = c + d;
	b = ROR24( b ^ c );
	a = a + b + m[ blake2b_sigma[r][2*i+1] ];
	d = ROR16( d ^ a );
	c = c + d;
	b = ROR2( b ^ c, 63);
}

#define ROUND(r) \
	G(r, 0, v[0], v[4], v[ 8], v[12], m); \
	G(r, 1, v[1], v[5], v[ 9], v[13], m); \
	G(r, 2, v[2], v[6], v[10], v[14], m); \
	G(r, 3, v[3], v[7], v[11], v[15], m); \
	G(r, 4, v[0], v[5], v[10], v[15], m); \
	G(r, 5, v[1], v[6], v[11], v[12], m); \
	G(r, 6, v[2], v[7], v[ 8], v[13], m); \
	G(r, 7, v[3], v[4], v[ 9], v[14], m);

__global__ __launch_bounds__(512,1)
void blake2b_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t target6)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	
	if(thread<threads){

		const uint32_t nonce = thread + startNonce;

		uint2 v[16];
		uint2 m[16];

		*(uint2x4*)&m[0] = *(uint2x4*)&c_data[0];
		*(uint2x4*)&m[4] = *(uint2x4*)&c_data[4];
		m[4].x = nonce;
		m[8] = c_data[8];
		m[9] = c_data[9];

		m[10] = m[11] = make_uint2(0,0);
		m[12] = m[13] = m[14] = m[15] = make_uint2(0,0);

		#pragma unroll 4
		for(uint32_t i=0;i<16;i+=4){
			*(uint2x4*)&v[i] = *(uint2x4*)&c_v[i];
		}

		v[ 2] = v[ 2] + m[4];
		v[14] = SWAPUINT2( v[14] ^ v[2] );
		v[10] = v[10] + v[14];
		v[ 6] = ROR24( v[ 6] ^ v[10] );
		v[ 2] = v[ 2] + v[ 6] + m[ 5];
		v[14] = ROR16( v[14] ^ v[ 2] );
		v[10] = v[10] + v[14];
		v[ 6] = ROR2( v[ 6] ^ v[10], 63);


		v[10] = v[10] + v[15];
		v[ 5] = ROR24( v[ 5] ^ v[10] );
		v[ 0] = v[ 0] + v[ 5];
		v[15] = ROR16(v[15] ^ v[0]);
		v[10] = v[10] + v[15];
		v[ 5] = ROR2( v[ 5] ^ v[10], 63);
		
		G(0, 5, v[1], v[6], v[11], v[12], m);
		G(0, 6, v[2], v[7], v[ 8], v[13], m);
		G(0, 7, v[3], v[4], v[ 9], v[14], m);
		ROUND( 1 );
		ROUND( 2 );
		ROUND( 3 );
		ROUND( 4 );
		ROUND( 5 );
		ROUND( 6 );
		ROUND( 7 );
		ROUND( 8 );
		ROUND( 9 );
		ROUND( 10 );
//		ROUND_F( 11 );
		G(11, 0, v[0], v[4], v[ 8], v[12], m);
		G(11, 1, v[1], v[5], v[ 9], v[13], m);
		G(11, 2, v[2], v[6], v[10], v[14], m);
		G(11, 3, v[3], v[7], v[11], v[15], m);
//		G(11, 4, v[0], v[5], v[10], v[15], m);
		v[ 0] = v[ 0] + v[ 5] + m[ 1];
		v[15] = SWAPUINT2( v[15] ^ v[0] );
		v[10] = v[10] + v[15];
		v[ 5] = ROR24( v[ 5] ^ v[10] );
		v[ 0] = v[ 0] + v[ 5];
//		G(11, 5, v[1], v[6], v[11], v[12], m);

//		H(11, 6, v[2], v[7], v[ 8], v[13], m);
		v[ 2] = v[ 2] + v[ 7] + m[blake2b_sigma[11][12]];
		v[13] = SWAPUINT2( v[13] ^ v[2]);
		v[ 8] = v[ 8] + v[13];
		v[ 7] = ROR24( v[7] ^ v[8] );
		v[ 2] = v[ 2] + v[ 7] + m[blake2b_sigma[11][13]];
		v[13] = ROR16( v[13] ^ v[2] );
		v[ 8] = v[ 8] + v[13];
		
		if (xor3x(v[8].x, v[0].x, 0xf2bdc928) == 0){
			if (cuda_swab32(0x6a09e667 ^ v[0].y ^ v[8].y ) <= target6) {
				uint32_t tmp = atomicExch(&resNonce[0], nonce);
				if (tmp != UINT32_MAX)
					resNonce[1] = tmp;
			}
		}
	}
}

__host__
uint32_t blake2b_hash_cuda(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint32_t target6, uint32_t &secNonce)
{
	uint32_t resNonces[NBN] = { UINT32_MAX, UINT32_MAX };
	uint32_t result = UINT32_MAX;

	if (hipSuccess == hipMemcpy(resNonces, d_resNonces[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = resNonces[0];
		secNonce = resNonces[1];
		if (secNonce == result) secNonce = UINT32_MAX;
	}
	return result;
}

__host__
void blake2b_setBlock(uint32_t *data)
{
	uint64_t v[16] = {
		0x6a09e667f2bdc928, 0xbb67ae8584caa73b, 0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1, 0x510e527fade682d1, 0x9b05688c2b3e6c1f, 0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
		0x6a09e667f3bcc908, 0xbb67ae8584caa73b, 0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1, 0x510e527fade68281, 0x9b05688c2b3e6c1f, 0xe07c265404be4294, 0x5be0cd19137e2179
	};
	uint64_t m[16];
	memcpy(m,data,80);
	memset(&m[10],0x00,6*sizeof(uint64_t));

	v[ 0]+= v[ 4] + m[ 0];
	v[12] = ROTR64(v[12] ^ v[ 0],32);
	v[ 8]+= v[12];
	v[ 4] = ROTR64(v[ 4] ^ v[ 8],24);
	v[ 0]+= v[ 4] + m[ 1];
	v[12] = ROTR64(v[12] ^ v[ 0],16);
	v[ 8]+= v[12];
	v[ 4] = ROTR64(v[ 4] ^ v[ 8],63);

	v[ 1] = v[ 1] + v[ 5] + m[ 2];
	v[13] = ROTR64( v[13] ^ v[1],32);
	v[ 9] = v[ 9] + v[13];
	v[ 5] = ROTR64( v[5] ^ v[9],24);
	v[ 1] = v[ 1] + v[ 5] + m[ 3];
	v[13] = ROTR64( v[13] ^ v[1],16);
	v[ 9] = v[ 9] + v[13];
	v[ 5] = ROTR64( v[5] ^ v[9], 63);

	v[ 2] = v[ 2] + v[ 6];

	v[ 3] = v[ 3] + v[ 7] + m[6];
	v[15] = ROTR64( v[15] ^ v[3] ,32);
	v[11] = v[11] + v[15];
	v[ 7] = ROTR64( v[7] ^ v[11] ,24);
	v[ 3] = v[ 3] + v[ 7] + m[7];
	v[15] = ROTR64( v[15] ^ v[3] ,16);
	v[11] = v[11] + v[15];
	v[ 7] = ROTR64( v[7] ^ v[11], 63);

	v[ 0] = v[ 0] + v[ 5] + m[8];
	v[15] = ROTR64( v[15] ^ v[0] ,32);
	v[ 0] = v[ 0] + m[9];
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, 80, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_v), v, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

static bool init[MAX_GPUS] = { 0 };

int scanhash_sia(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done){

	int dev_id = device_map[thr_id];
	
	uint32_t _ALIGN(64) hash[8];
	uint32_t _ALIGN(64) vhashcpu[8];
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t Htarg = ptarget[7];
	const uint32_t first_nonce = pdata[8];

	int intensity = (device_sm[dev_id] > 500)?29:28;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO,dev_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		
		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonces[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonces[thr_id], NBN * sizeof(uint32_t)), -1);

		init[thr_id] = true;
	}
	const dim3 grid((throughput + TPB-1)/TPB);
	const dim3 block(TPB);

	memcpy(endiandata, pdata, 80);
	endiandata[11] = 0; // nbits

	blake2b_setBlock(endiandata);

	hipMemset(d_resNonces[thr_id], 0xff, NBN*sizeof(uint32_t));

	do {
		blake2b_gpu_hash <<<grid, block, 8>>> (throughput, pdata[8], d_resNonces[thr_id], ptarget[6]);
		
		hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);
		
		if (h_resNonces[thr_id][0] != UINT32_MAX){
			int res = 0;
			endiandata[8] = h_resNonces[thr_id][0];
			blake2b_hash(hash, endiandata);
			// sia hash target is reversed (start of hash)
			swab256(vhashcpu, hash);
			if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget)) {
				work_set_target_ratio(work, vhashcpu);
				*hashes_done = pdata[8] - first_nonce + throughput +1;
				work->nonces[0] = h_resNonces[thr_id][0];
				pdata[8] = h_resNonces[thr_id][0];
				res=1;
				if (h_resNonces[thr_id][1] != UINT32_MAX) {
					endiandata[8] = h_resNonces[thr_id][1];
					blake2b_hash(hash, endiandata);
//					if(!opt_quiet)
//						gpulog(LOG_BLUE, dev_id, "Found 2nd nonce: %08x", h_resNonces[thr_id][1]);
					swab256(vhashcpu, hash);
					work->nonces[1] = h_resNonces[thr_id][1];
					pdata[21] = h_resNonces[thr_id][1];
					if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio[0]) {
						work_set_target_ratio(work, vhashcpu);
						xchg(work->nonces[0], work->nonces[1]);
						xchg(pdata[8], pdata[21]);
					}
					res=2;
				}
				return res;
			}
		}

		pdata[8] += throughput;

	}while(!work_restart[thr_id].restart && ((uint64_t)max_nonce > (uint64_t)throughput + pdata[8]));

	*hashes_done = pdata[8] - first_nonce +1;

	return 0;
}

// cleanup
extern "C" void free_sia(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_resNonces[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

// ---- SIA LONGPOLL --------------------------------------------------------------------------------

struct data_buffer {
	void *buf;
	size_t len;
};

extern void calc_network_diff(struct work *work);


size_t sia_data_cb(const void *ptr, size_t size, size_t nmemb, void *user_data){
	struct data_buffer *db = (struct data_buffer *)user_data;
	size_t len = size * nmemb;
	size_t oldlen, newlen;
	void *newmem;
	static const uchar zero = 0;

	oldlen = db->len;
	newlen = oldlen + len;

	newmem = realloc(db->buf, newlen + 1);
	if (!newmem)
		return 0;

	db->buf = newmem;
	db->len = newlen;
	memcpy((char*)db->buf + oldlen, ptr, len);
	memcpy((char*)db->buf + newlen, &zero, 1);	/* null terminate */

	return len;
}

char* sia_getheader(CURL *curl, struct pool_infos *pool)
{
	char curl_err_str[CURL_ERROR_SIZE] = { 0 };
	struct data_buffer all_data = { 0 };
	struct curl_slist *headers = NULL;
	char data[256] = { 0 };
	char url[512];

	// nanopool
	snprintf(url, 512, "%s/miner/header?address=%s&worker=%s", //&longpoll
		pool->url, pool->user, pool->pass);

	if (opt_protocol)
		curl_easy_setopt(curl, CURLOPT_VERBOSE, 1);
	curl_easy_setopt(curl, CURLOPT_URL, url);
	curl_easy_setopt(curl, CURLOPT_POST, 0);
	curl_easy_setopt(curl, CURLOPT_ENCODING, "");
	curl_easy_setopt(curl, CURLOPT_FAILONERROR, 0);
	curl_easy_setopt(curl, CURLOPT_FOLLOWLOCATION, 1);
	curl_easy_setopt(curl, CURLOPT_TCP_NODELAY, 1);
	curl_easy_setopt(curl, CURLOPT_TIMEOUT, opt_timeout);
	curl_easy_setopt(curl, CURLOPT_NOSIGNAL, 1);
	curl_easy_setopt(curl, CURLOPT_ERRORBUFFER, curl_err_str);
	curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, sia_data_cb);
	curl_easy_setopt(curl, CURLOPT_WRITEDATA, &all_data);

	headers = curl_slist_append(headers, "Accept: application/octet-stream");
	headers = curl_slist_append(headers, "Expect:"); // disable Expect hdr
	headers = curl_slist_append(headers, "User-Agent: Sia-Agent"); // required for now
//	headers = curl_slist_append(headers, "User-Agent: " USER_AGENT);
//	headers = curl_slist_append(headers, "X-Mining-Extensions: longpoll");

	curl_easy_setopt(curl, CURLOPT_HTTPHEADER, headers);

	int rc = curl_easy_perform(curl);
	if (rc && strlen(curl_err_str)) {
		applog(LOG_WARNING, "%s", curl_err_str);
	}

	if (all_data.len >= 112)
		cbin2hex(data, (const char*) all_data.buf, 112);
	if (opt_protocol || all_data.len != 112)
		applog(LOG_DEBUG, "received %d bytes: %s", (int) all_data.len, data);

	curl_slist_free_all(headers);

	return rc == 0 && all_data.len ? strdup(data) : NULL;
}

bool sia_work_decode(const char *hexdata, struct work *work)
{
	uint8_t target[32];
	if (!work) return false;

	hex2bin((uchar*)target, &hexdata[0], 32);
	swab256(work->target, target);
	work->targetdiff = target_to_diff(work->target);

	hex2bin((uchar*)work->data, &hexdata[64], 80);
	// high 16 bits of the 64 bits nonce
	work->data[9] = rand() << 16;

	// use work ntime as job id
	cbin2hex(work->job_id, (const char*)&work->data[10], 4);
	calc_network_diff(work);

	if (stratum_diff != work->targetdiff) {
		stratum_diff = work->targetdiff;
		applog(LOG_WARNING, "Pool diff set to %g", stratum_diff);
	}

	return true;
}

extern int share_result(int result, int pooln, double sharediff, const char *reason);

bool sia_submit(CURL *curl, struct pool_infos *pool, struct work *work){

	char curl_err_str[CURL_ERROR_SIZE] = { 0 };
	struct data_buffer all_data = { 0 };
	struct curl_slist *headers = NULL;
	char buf[256] = { 0 };
	char url[512];

	if (opt_protocol)
		applog_hex(work->data, 80);
	//applog_hex(&work->data[8], 16);
	//applog_hex(&work->data[10], 4);

	// nanopool
	snprintf(url, 512, "%s/miner/header?address=%s&worker=%s",
		pool->url, pool->user, pool->pass);

	if (opt_protocol)
		curl_easy_setopt(curl, CURLOPT_VERBOSE, 1);
	curl_easy_setopt(curl, CURLOPT_URL, url);
	curl_easy_setopt(curl, CURLOPT_ENCODING, "");
	curl_easy_setopt(curl, CURLOPT_FAILONERROR, 0);
	curl_easy_setopt(curl, CURLOPT_NOSIGNAL, 1);
	curl_easy_setopt(curl, CURLOPT_TCP_NODELAY, 1);
	curl_easy_setopt(curl, CURLOPT_ERRORBUFFER, curl_err_str);
	curl_easy_setopt(curl, CURLOPT_FOLLOWLOCATION, 1);
	curl_easy_setopt(curl, CURLOPT_TIMEOUT, 10);

	curl_easy_setopt(curl, CURLOPT_WRITEDATA, &all_data);
	curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, sia_data_cb);

	memcpy(buf, work->data, 80);
	curl_easy_setopt(curl, CURLOPT_POST, 1);
	curl_easy_setopt(curl, CURLOPT_POSTFIELDSIZE, 80);
	curl_easy_setopt(curl, CURLOPT_POSTFIELDS, (void*) buf);

//	headers = curl_slist_append(headers, "Content-Type: application/octet-stream");
//	headers = curl_slist_append(headers, "Content-Length: 80");
	headers = curl_slist_append(headers, "Accept:"); // disable Accept hdr
	headers = curl_slist_append(headers, "Expect:"); // disable Expect hdr
	headers = curl_slist_append(headers, "User-Agent: Sia-Agent");
//	headers = curl_slist_append(headers, "User-Agent: " USER_AGENT);
	curl_easy_setopt(curl, CURLOPT_HTTPHEADER, headers);

	int res = curl_easy_perform(curl) == 0;
	long errcode;
	CURLcode c = curl_easy_getinfo(curl, CURLINFO_RESPONSE_CODE, &errcode);
	if (errcode != 204) {
		if (strlen(curl_err_str))
			applog(LOG_ERR, "submit err %ld %s", errcode, curl_err_str);
		res = 0;
	}
	share_result(res, work->pooln, work->sharediff[0], res ? NULL : (char*) all_data.buf);

	curl_slist_free_all(headers);
	return true;
}
