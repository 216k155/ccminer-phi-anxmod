#include "hip/hip_runtime.h"
/***************************************************************************************************
 * SIMD512 SM3+ CUDA IMPLEMENTATION (require cuda_x11_simd512_func.cuh)
 */

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_vectors.h"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#define TPB50_1 128
#define TPB50_2 128
#define TPB52_1 128
#define TPB52_2 128

static uint4 *d_temp4[MAX_GPUS];
#include "cuda_x11_simd512_func.cuh"

__global__ 
#if __CUDA_ARCH__ > 500
__launch_bounds__(TPB52_2,1)
#else
__launch_bounds__(TPB50_2,4)
#endif
static void x11_simd512_gpu_compress_64_maxwell(uint32_t threads, uint32_t *g_hash,const uint4 *const __restrict__ g_fft4)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t thr_offset = thread << 6; // thr_id * 128 (je zwei elemente)
	uint32_t IV[32];
	if (thread < threads){

		uint32_t *Hash = &g_hash[thread<<4];
//		Compression1(Hash, thread, g_fft4, g_state);
		uint32_t A[32];

		*(uint2x4*)&IV[ 0] = *(uint2x4*)&c_IV_512[ 0];
		*(uint2x4*)&IV[ 8] = *(uint2x4*)&c_IV_512[ 8];
		*(uint2x4*)&IV[16] = *(uint2x4*)&c_IV_512[16];
		*(uint2x4*)&IV[24] = *(uint2x4*)&c_IV_512[24];

		*(uint2x4*)&A[ 0] = __ldg4((uint2x4*)&Hash[ 0]);
		*(uint2x4*)&A[ 8] = __ldg4((uint2x4*)&Hash[ 8]);

		#pragma unroll 16
		for(uint32_t i=0;i<16;i++)
			A[ i] = A[ i] ^ IV[ i];

		#pragma unroll 16
		for(uint32_t i=16;i<32;i++)
			A[ i] = IV[ i];

		Round8(A, thr_offset, g_fft4);
		
		STEP8_IF(&IV[ 0],32, 4,13,&A[ 0],&A[ 8],&A[16],&A[24]);
		STEP8_IF(&IV[ 8],33,13,10,&A[24],&A[ 0],&A[ 8],&A[16]);
		STEP8_IF(&IV[16],34,10,25,&A[16],&A[24],&A[ 0],&A[ 8]);
		STEP8_IF(&IV[24],35,25, 4,&A[ 8],&A[16],&A[24],&A[ 0]);

		#pragma unroll 32
		for(uint32_t i=0;i<32;i++){
			IV[ i] = A[ i];
		}
		
		A[ 0] ^= 512;

		Round8_0_final(A, 3,23,17,27);
		Round8_1_final(A,28,19,22, 7);
		Round8_2_final(A,29, 9,15, 5);
		Round8_3_final(A, 4,13,10,25);
		STEP8_IF(&IV[ 0],32, 4,13, &A[ 0], &A[ 8], &A[16], &A[24]);
		STEP8_IF(&IV[ 8],33,13,10, &A[24], &A[ 0], &A[ 8], &A[16]);
		STEP8_IF(&IV[16],34,10,25, &A[16], &A[24], &A[ 0], &A[ 8]);
		STEP8_IF(&IV[24],35,25, 4, &A[ 8], &A[16], &A[24], &A[ 0]);

		*(uint2x4*)&Hash[ 0] = *(uint2x4*)&A[ 0];
		*(uint2x4*)&Hash[ 8] = *(uint2x4*)&A[ 8];
	}
}

__host__
void x11_simd512_cpu_init(int thr_id, uint32_t threads){
	hipMalloc(&d_temp4[thr_id], 64*sizeof(uint4)*threads);
}

__host__
void x11_simd512_cpu_free(int thr_id){
	hipFree(d_temp4[thr_id]);
}

__host__
void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash){

	int dev_id = device_map[thr_id];

	uint32_t tpb = TPB52_1;
	if (device_sm[dev_id] <= 500) tpb = TPB50_1;
	const dim3 grid1((8*threads + tpb - 1) / tpb);
	const dim3 block1(tpb);

	tpb = TPB52_2;
	if (device_sm[dev_id] <= 500) tpb = TPB50_2;
	const dim3 grid2((threads + tpb - 1) / tpb);
	const dim3 block2(tpb);
	
	x11_simd512_gpu_expand_64 <<<grid1, block1>>> (threads, d_hash, d_temp4[thr_id]);
	x11_simd512_gpu_compress_64_maxwell <<< grid2, block2 >>> (threads, d_hash, d_temp4[thr_id]);
}
