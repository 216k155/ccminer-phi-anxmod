#include "hip/hip_runtime.h"
/*
 * Quick Hamsi-512 for X13
 * by tsiv - 2014
 * Provos Alexis - 2016
 */

/*
 * X13 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014-2016  phm, Provos Alexis
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com> (fugue512 implementation / 2014)
 * @author   Provos Alexis (Merged hamsi512 and fugue512, Applied partial shared Mem utilization under Cuda7.5/ 2016)
 */
#include "miner.h"
#include "cuda_helper.h"
#include "cuda_vectors.h"

//-------HAMSI MACROS--------------------------------------------

static __constant__ const uint32_t d_alpha_n[] = {
	0xff00f0f0, 0xccccaaaa, 0xf0f0cccc, 0xff00aaaa, 0xccccaaaa, 0xf0f0ff00, 0xaaaacccc, 0xf0f0ff00,	0xf0f0cccc, 0xaaaaff00, 0xccccff00, 0xaaaaf0f0, 0xaaaaf0f0, 0xff00cccc, 0xccccf0f0, 0xff00aaaa,
	0xccccaaaa, 0xff00f0f0, 0xff00aaaa, 0xf0f0cccc, 0xf0f0ff00, 0xccccaaaa, 0xf0f0ff00, 0xaaaacccc,	0xaaaaff00, 0xf0f0cccc, 0xaaaaf0f0, 0xccccff00, 0xff00cccc, 0xaaaaf0f0, 0xff00aaaa, 0xccccf0f0
};

static __constant__ const uint32_t d_alpha_f[] = {
	0xcaf9639c, 0x0ff0f9c0, 0x639c0ff0, 0xcaf9f9c0, 0x0ff0f9c0, 0x639ccaf9, 0xf9c00ff0, 0x639ccaf9,	0x639c0ff0, 0xf9c0caf9, 0x0ff0caf9, 0xf9c0639c, 0xf9c0639c, 0xcaf90ff0, 0x0ff0639c, 0xcaf9f9c0,
	0x0ff0f9c0, 0xcaf9639c, 0xcaf9f9c0, 0x639c0ff0, 0x639ccaf9, 0x0ff0f9c0, 0x639ccaf9, 0xf9c00ff0,	0xf9c0caf9, 0x639c0ff0, 0xf9c0639c, 0x0ff0caf9, 0xcaf90ff0, 0xf9c0639c, 0xcaf9f9c0, 0x0ff0639c
};

static __constant__ const uint32_t c_c[] = {
		0x73746565, 0x6c706172, 0x6b204172, 0x656e6265, 0x72672031, 0x302c2062, 0x75732032, 0x3434362c,
		0x20422d33, 0x30303120, 0x4c657576, 0x656e2d48, 0x65766572, 0x6c65652c, 0x2042656c, 0x6769756d
};

static __constant__ const uint32_t d_T512[1024] = {
	0xef0b0270, 0x3afd0000, 0x5dae0000, 0x69490000, 0x9b0f3c06, 0x4405b5f9, 0x66140a51, 0x924f5d0a, 0xc96b0030, 0xe7250000, 0x2f840000, 0x264f0000, 0x08695bf9, 0x6dfcf137, 0x509f6984, 0x9e69af68,
	0xc96b0030, 0xe7250000, 0x2f840000, 0x264f0000, 0x08695bf9, 0x6dfcf137, 0x509f6984, 0x9e69af68, 0x26600240, 0xddd80000, 0x722a0000, 0x4f060000, 0x936667ff, 0x29f944ce, 0x368b63d5, 0x0c26f262,
	0x145a3c00, 0xb9e90000, 0x61270000, 0xf1610000, 0xce613d6c, 0xb0493d78, 0x47a96720, 0xe18e24c5, 0x23671400, 0xc8b90000, 0xf4c70000, 0xfb750000, 0x73cd2465, 0xf8a6a549, 0x02c40a3f, 0xdc24e61f,
	0x23671400, 0xc8b90000, 0xf4c70000, 0xfb750000, 0x73cd2465, 0xf8a6a549, 0x02c40a3f, 0xdc24e61f, 0x373d2800, 0x71500000, 0x95e00000, 0x0a140000, 0xbdac1909, 0x48ef9831, 0x456d6d1f, 0x3daac2da,
	0x54285c00, 0xeaed0000, 0xc5d60000, 0xa1c50000, 0xb3a26770, 0x94a5c4e1, 0x6bb0419d, 0x551b3782, 0x9cbb1800, 0xb0d30000, 0x92510000, 0xed930000, 0x593a4345, 0xe114d5f4, 0x430633da, 0x78cace29,
	0x9cbb1800, 0xb0d30000, 0x92510000, 0xed930000, 0x593a4345, 0xe114d5f4, 0x430633da, 0x78cace29, 0xc8934400, 0x5a3e0000, 0x57870000, 0x4c560000, 0xea982435, 0x75b11115, 0x28b67247, 0x2dd1f9ab,
	0x29449c00, 0x64e70000, 0xf24b0000, 0xc2f30000, 0x0ede4e8f, 0x56c23745, 0xf3e04259, 0x8d0d9ec4, 0x466d0c00, 0x08620000, 0xdd5d0000, 0xbadd0000, 0x6a927942, 0x441f2b93, 0x218ace6f, 0xbf2c0be2,
	0x466d0c00, 0x08620000, 0xdd5d0000, 0xbadd0000, 0x6a927942, 0x441f2b93, 0x218ace6f, 0xbf2c0be2, 0x6f299000, 0x6c850000, 0x2f160000, 0x782e0000, 0x644c37cd, 0x12dd1cd6, 0xd26a8c36, 0x32219526,
	0xf6800005, 0x3443c000, 0x24070000, 0x8f3d0000, 0x21373bfb, 0x0ab8d5ae, 0xcdc58b19, 0xd795ba31, 0xa67f0001, 0x71378000, 0x19fc0000, 0x96db0000, 0x3a8b6dfd, 0xebcaaef3, 0x2c6d478f, 0xac8e6c88,
	0xa67f0001, 0x71378000, 0x19fc0000, 0x96db0000, 0x3a8b6dfd, 0xebcaaef3, 0x2c6d478f, 0xac8e6c88, 0x50ff0004, 0x45744000, 0x3dfb0000, 0x19e60000, 0x1bbc5606, 0xe1727b5d, 0xe1a8cc96, 0x7b1bd6b9,
	0xf7750009, 0xcf3cc000, 0xc3d60000, 0x04920000, 0x029519a9, 0xf8e836ba, 0x7a87f14e, 0x9e16981a, 0xd46a0000, 0x8dc8c000, 0xa5af0000, 0x4a290000, 0xfc4e427a, 0xc9b4866c, 0x98369604, 0xf746c320,
	0xd46a0000, 0x8dc8c000, 0xa5af0000, 0x4a290000, 0xfc4e427a, 0xc9b4866c, 0x98369604, 0xf746c320, 0x231f0009, 0x42f40000, 0x66790000, 0x4ebb0000, 0xfedb5bd3, 0x315cb0d6, 0xe2b1674a, 0x69505b3a,
	0x774400f0, 0xf15a0000, 0xf5b20000, 0x34140000, 0x89377e8c, 0x5a8bec25, 0x0bc3cd1e, 0xcf3775cb, 0xf46c0050, 0x96180000, 0x14a50000, 0x031f0000, 0x42947eb8, 0x66bf7e19, 0x9ca470d2, 0x8a341574,
	0xf46c0050, 0x96180000, 0x14a50000, 0x031f0000, 0x42947eb8, 0x66bf7e19, 0x9ca470d2, 0x8a341574, 0x832800a0, 0x67420000, 0xe1170000, 0x370b0000, 0xcba30034, 0x3c34923c, 0x9767bdcc, 0x450360bf,
	0xe8870170, 0x9d720000, 0x12db0000, 0xd4220000, 0xf2886b27, 0xa921e543, 0x4ef8b518, 0x618813b1, 0xb4370060, 0x0c4c0000, 0x56c20000, 0x5cae0000, 0x94541f3f, 0x3b3ef825, 0x1b365f3d, 0xf3d45758,
	0xb4370060, 0x0c4c0000, 0x56c20000, 0x5cae0000, 0x94541f3f, 0x3b3ef825, 0x1b365f3d, 0xf3d45758, 0x5cb00110, 0x913e0000, 0x44190000, 0x888c0000, 0x66dc7418, 0x921f1d66, 0x55ceea25, 0x925c44e9,
	0x0c720000, 0x49e50f00, 0x42790000, 0x5cea0000, 0x33aa301a, 0x15822514, 0x95a34b7b, 0xb44b0090, 0xfe220000, 0xa7580500, 0x25d10000, 0xf7600000, 0x893178da, 0x1fd4f860, 0x4ed0a315, 0xa123ff9f,
	0xfe220000, 0xa7580500, 0x25d10000, 0xf7600000, 0x893178da, 0x1fd4f860, 0x4ed0a315, 0xa123ff9f, 0xf2500000, 0xeebd0a00, 0x67a80000, 0xab8a0000, 0xba9b48c0, 0x0a56dd74, 0xdb73e86e, 0x1568ff0f,
	0x45180000, 0xa5b51700, 0xf96a0000, 0x3b480000, 0x1ecc142c, 0x231395d6, 0x16bca6b0, 0xdf33f4df, 0xb83d0000, 0x16710600, 0x379a0000, 0xf5b10000, 0x228161ac, 0xae48f145, 0x66241616, 0xc5c1eb3e,
	0xb83d0000, 0x16710600, 0x379a0000, 0xf5b10000, 0x228161ac, 0xae48f145, 0x66241616, 0xc5c1eb3e, 0xfd250000, 0xb3c41100, 0xcef00000, 0xcef90000, 0x3c4d7580, 0x8d5b6493, 0x7098b0a6, 0x1af21fe1,
	0x75a40000, 0xc28b2700, 0x94a40000, 0x90f50000, 0xfb7857e0, 0x49ce0bae, 0x1767c483, 0xaedf667e, 0xd1660000, 0x1bbc0300, 0x9eec0000, 0xf6940000, 0x03024527, 0xcf70fcf2, 0xb4431b17, 0x857f3c2b,
	0xd1660000, 0x1bbc0300, 0x9eec0000, 0xf6940000, 0x03024527, 0xcf70fcf2, 0xb4431b17, 0x857f3c2b, 0xa4c20000, 0xd9372400, 0x0a480000, 0x66610000, 0xf87a12c7, 0x86bef75c, 0xa324df94, 0x2ba05a55,
	0x75c90003, 0x0e10c000, 0xd1200000, 0xbaea0000, 0x8bc42f3e, 0x8758b757, 0xbb28761d, 0x00b72e2b, 0xeecf0001, 0x6f564000, 0xf33e0000, 0xa79e0000, 0xbdb57219, 0xb711ebc5, 0x4a3b40ba, 0xfeabf254,
	0xeecf0001, 0x6f564000, 0xf33e0000, 0xa79e0000, 0xbdb57219, 0xb711ebc5, 0x4a3b40ba, 0xfeabf254, 0x9b060002, 0x61468000, 0x221e0000, 0x1d740000, 0x36715d27, 0x30495c92, 0xf11336a7, 0xfe1cdc7f,
	0x86790000, 0x3f390002, 0xe19ae000, 0x98560000, 0x9565670e, 0x4e88c8ea, 0xd3dd4944, 0x161ddab9, 0x30b70000, 0xe5d00000, 0xf4f46000, 0x42c40000, 0x63b83d6a, 0x78ba9460, 0x21afa1ea, 0xb0a51834,
	0x30b70000, 0xe5d00000, 0xf4f46000, 0x42c40000, 0x63b83d6a, 0x78ba9460, 0x21afa1ea, 0xb0a51834, 0xb6ce0000, 0xdae90002, 0x156e8000, 0xda920000, 0xf6dd5a64, 0x36325c8a, 0xf272e8ae, 0xa6b8c28d,
	0x14190000, 0x23ca003c, 0x50df0000, 0x44b60000, 0x1b6c67b0, 0x3cf3ac75, 0x61e610b0, 0xdbcadb80, 0xe3430000, 0x3a4e0014, 0xf2c60000, 0xaa4e0000, 0xdb1e42a6, 0x256bbe15, 0x123db156, 0x3a4e99d7,
	0xe3430000, 0x3a4e0014, 0xf2c60000, 0xaa4e0000, 0xdb1e42a6, 0x256bbe15, 0x123db156, 0x3a4e99d7, 0xf75a0000, 0x19840028, 0xa2190000, 0xeef80000, 0xc0722516, 0x19981260, 0x73dba1e6, 0xe1844257,
	0x54500000, 0x0671005c, 0x25ae0000, 0x6a1e0000, 0x2ea54edf, 0x664e8512, 0xbfba18c3, 0x7e715d17, 0xbc8d0000, 0xfc3b0018, 0x19830000, 0xd10b0000, 0xae1878c4, 0x42a69856, 0x0012da37, 0x2c3b504e,
	0xbc8d0000, 0xfc3b0018, 0x19830000, 0xd10b0000, 0xae1878c4, 0x42a69856, 0x0012da37, 0x2c3b504e, 0xe8dd0000, 0xfa4a0044, 0x3c2d0000, 0xbb150000, 0x80bd361b, 0x24e81d44, 0xbfa8c2f4, 0x524a0d59,
	0x69510000, 0xd4e1009c, 0xc3230000, 0xac2f0000, 0xe4950bae, 0xcea415dc, 0x87ec287c, 0xbce1a3ce, 0xc6730000, 0xaf8d000c, 0xa4c10000, 0x218d0000, 0x23111587, 0x7913512f, 0x1d28ac88, 0x378dd173,
	0xc6730000, 0xaf8d000c, 0xa4c10000, 0x218d0000, 0x23111587, 0x7913512f, 0x1d28ac88, 0x378dd173, 0xaf220000, 0x7b6c0090, 0x67e20000, 0x8da20000, 0xc7841e29, 0xb7b744f3, 0x9ac484f4, 0x8b6c72bd,
	0xcc140000, 0xa5630000, 0x5ab90780, 0x3b500000, 0x4bd013ff, 0x879b3418, 0x694348c1, 0xca5a87fe, 0x819e0000, 0xec570000, 0x66320280, 0x95f30000, 0x5da92802, 0x48f43cbc, 0xe65aa22d, 0x8e67b7fa,
	0x819e0000, 0xec570000, 0x66320280, 0x95f30000, 0x5da92802, 0x48f43cbc, 0xe65aa22d, 0x8e67b7fa, 0x4d8a0000, 0x49340000, 0x3c8b0500, 0xaea30000, 0x16793bfd, 0xcf6f08a4, 0x8f19eaec, 0x443d3004,
	0x78230000, 0x12fc0000, 0xa93a0b80, 0x90a50000, 0x713e2879, 0x7ee98924, 0xf08ca062, 0x636f8bab, 0x02af0000, 0xb7280000, 0xba1c0300, 0x56980000, 0xba8d45d3, 0x8048c667, 0xa95c149a, 0xf4f6ea7b,
	0x02af0000, 0xb7280000, 0xba1c0300, 0x56980000, 0xba8d45d3, 0x8048c667, 0xa95c149a, 0xf4f6ea7b, 0x7a8c0000, 0xa5d40000, 0x13260880, 0xc63d0000, 0xcbb36daa, 0xfea14f43, 0x59d0b4f8, 0x979961d0,
	0xac480000, 0x1ba60000, 0x45fb1380, 0x03430000, 0x5a85316a, 0x1fb250b6, 0xfe72c7fe, 0x91e478f6, 0x1e4e0000, 0xdecf0000, 0x6df80180, 0x77240000, 0xec47079e, 0xf4a0694e, 0xcda31812, 0x98aa496e,
	0x1e4e0000, 0xdecf0000, 0x6df80180, 0x77240000, 0xec47079e, 0xf4a0694e, 0xcda31812, 0x98aa496e, 0xb2060000, 0xc5690000, 0x28031200, 0x74670000, 0xb6c236f4, 0xeb1239f8, 0x33d1dfec, 0x094e3198,
	0xaec30000, 0x9c4f0001, 0x79d1e000, 0x2c150000, 0x45cc75b3, 0x6650b736, 0xab92f78f, 0xa312567b, 0xdb250000, 0x09290000, 0x49aac000, 0x81e10000, 0xcafe6b59, 0x42793431, 0x43566b76, 0xe86cba2e,
	0xdb250000, 0x09290000, 0x49aac000, 0x81e10000, 0xcafe6b59, 0x42793431, 0x43566b76, 0xe86cba2e, 0x75e60000, 0x95660001, 0x307b2000, 0xadf40000, 0x8f321eea, 0x24298307, 0xe8c49cf9, 0x4b7eec55,
	0x58430000, 0x807e0000, 0x78330001, 0xc66b3800, 0xe7375cdc, 0x79ad3fdd, 0xac73fe6f, 0x3a4479b1, 0x1d5a0000, 0x2b720000, 0x488d0000, 0xaf611800, 0x25cb2ec5, 0xc879bfd0, 0x81a20429, 0x1e7536a6,
	0x1d5a0000, 0x2b720000, 0x488d0000, 0xaf611800, 0x25cb2ec5, 0xc879bfd0, 0x81a20429, 0x1e7536a6, 0x45190000, 0xab0c0000, 0x30be0001, 0x690a2000, 0xc2fc7219, 0xb1d4800d, 0x2dd1fa46, 0x24314f17,
	0xa53b0000, 0x14260000, 0x4e30001e, 0x7cae0000, 0x8f9e0dd5, 0x78dfaa3d, 0xf73168d8, 0x0b1b4946, 0x07ed0000, 0xb2500000, 0x8774000a, 0x970d0000, 0x437223ae, 0x48c76ea4, 0xf4786222, 0x9075b1ce,
	0x07ed0000, 0xb2500000, 0x8774000a, 0x970d0000, 0x437223ae, 0x48c76ea4, 0xf4786222, 0x9075b1ce, 0xa2d60000, 0xa6760000, 0xc9440014, 0xeba30000, 0xccec2e7b, 0x3018c499, 0x03490afa, 0x9b6ef888,
	0x88980000, 0x1f940000, 0x7fcf002e, 0xfb4e0000, 0xf158079a, 0x61ae9167, 0xa895706c, 0xe6107494, 0x0bc20000, 0xdb630000, 0x7e88000c, 0x15860000, 0x91fd48f3, 0x7581bb43, 0xf460449e, 0xd8b61463,
	0x0bc20000, 0xdb630000, 0x7e88000c, 0x15860000, 0x91fd48f3, 0x7581bb43, 0xf460449e, 0xd8b61463, 0x835a0000, 0xc4f70000, 0x01470022, 0xeec80000, 0x60a54f69, 0x142f2a24, 0x5cf534f2, 0x3ea660f7,
	0x52500000, 0x29540000, 0x6a61004e, 0xf0ff0000, 0x9a317eec, 0x452341ce, 0xcf568fe5, 0x5303130f, 0x538d0000, 0xa9fc0000, 0x9ef70006, 0x56ff0000, 0x0ae4004e, 0x92c5cdf9, 0xa9444018, 0x7f975691,
	0x538d0000, 0xa9fc0000, 0x9ef70006, 0x56ff0000, 0x0ae4004e, 0x92c5cdf9, 0xa9444018, 0x7f975691, 0x01dd0000, 0x80a80000, 0xf4960048, 0xa6000000, 0x90d57ea2, 0xd7e68c37, 0x6612cffd, 0x2c94459e,
	0xe6280000, 0x4c4b0000, 0xa8550000, 0xd3d002e0, 0xd86130b8, 0x98a7b0da, 0x289506b4, 0xd75a4897, 0xf0c50000, 0x59230000, 0x45820000, 0xe18d00c0, 0x3b6d0631, 0xc2ed5699, 0xcbe0fe1c, 0x56a7b19f,
	0xf0c50000, 0x59230000, 0x45820000, 0xe18d00c0, 0x3b6d0631, 0xc2ed5699, 0xcbe0fe1c, 0x56a7b19f, 0x16ed0000, 0x15680000, 0xedd70000, 0x325d0220, 0xe30c3689, 0x5a4ae643, 0xe375f8a8, 0x81fdf908,
	0xb4310000, 0x77330000, 0xb15d0000, 0x7fd004e0, 0x78a26138, 0xd116c35d, 0xd256d489, 0x4e6f74de, 0xe3060000, 0xbdc10000, 0x87130000, 0xbff20060, 0x2eba0a1a, 0x8db53751, 0x73c5ab06, 0x5bd61539,
	0xe3060000, 0xbdc10000, 0x87130000, 0xbff20060, 0x2eba0a1a, 0x8db53751, 0x73c5ab06, 0x5bd61539, 0x57370000, 0xcaf20000, 0x364e0000, 0xc0220480, 0x56186b22, 0x5ca3f40c, 0xa1937f8f, 0x15b961e7,
	0x02f20000, 0xa2810000, 0x873f0000, 0xe36c7800, 0x1e1d74ef, 0x073d2bd6, 0xc4c23237, 0x7f32259e, 0xbadd0000, 0x13ad0000, 0xb7e70000, 0xf7282800, 0xdf45144d, 0x361ac33a, 0xea5a8d14, 0x2a2c18f0,
	0xbadd0000, 0x13ad0000, 0xb7e70000, 0xf7282800, 0xdf45144d, 0x361ac33a, 0xea5a8d14, 0x2a2c18f0, 0xb82f0000, 0xb12c0000, 0x30d80000, 0x14445000, 0xc15860a2, 0x3127e8ec, 0x2e98bf23, 0x551e3d6e,
	0x1e6c0000, 0xc4420000, 0x8a2e0000, 0xbcb6b800, 0x2c4413b6, 0x8bfdd3da, 0x6a0c1bc8, 0xb99dc2eb, 0x92560000, 0x1eda0000, 0xea510000, 0xe8b13000, 0xa93556a5, 0xebfb6199, 0xb15c2254, 0x33c5244f,
	0x92560000, 0x1eda0000, 0xea510000, 0xe8b13000, 0xa93556a5, 0xebfb6199, 0xb15c2254, 0x33c5244f, 0x8c3a0000, 0xda980000, 0x607f0000, 0x54078800, 0x85714513, 0x6006b243, 0xdb50399c, 0x8a58e6a4,
	0x033d0000, 0x08b30000, 0xf33a0000, 0x3ac20007, 0x51298a50, 0x6b6e661f, 0x0ea5cfe3, 0xe6da7ffe, 0xa8da0000, 0x96be0000, 0x5c1d0000, 0x07da0002, 0x7d669583, 0x1f98708a, 0xbb668808, 0xda878000,
	0xa8da0000, 0x96be0000, 0x5c1d0000, 0x07da0002, 0x7d669583, 0x1f98708a, 0xbb668808, 0xda878000, 0xabe70000, 0x9e0d0000, 0xaf270000, 0x3d180005, 0x2c4f1fd3, 0x74f61695, 0xb5c347eb, 0x3c5dfffe,
	0x01930000, 0xe7820000, 0xedfb0000, 0xcf0c000b, 0x8dd08d58, 0xbca3b42e, 0x063661e1, 0x536f9e7b, 0x92280000, 0xdc850000, 0x57fa0000, 0x56dc0003, 0xbae92316, 0x5aefa30c, 0x90cef752, 0x7b1675d7,
	0x92280000, 0xdc850000, 0x57fa0000, 0x56dc0003, 0xbae92316, 0x5aefa30c, 0x90cef752, 0x7b1675d7, 0x93bb0000, 0x3b070000, 0xba010000, 0x99d00008, 0x3739ae4e, 0xe64c1722, 0x96f896b3, 0x2879ebac,
	0x5fa80000, 0x56030000, 0x43ae0000, 0x64f30013, 0x257e86bf, 0x1311944e, 0x541e95bf, 0x8ea4db69, 0x00440000, 0x7f480000, 0xda7c0000, 0x2a230001, 0x3badc9cc, 0xa9b69c87, 0x030a9e60, 0xbe0a679e,
	0x00440000, 0x7f480000, 0xda7c0000, 0x2a230001, 0x3badc9cc, 0xa9b69c87, 0x030a9e60, 0xbe0a679e, 0x5fec0000, 0x294b0000, 0x99d20000, 0x4ed00012, 0x1ed34f73, 0xbaa708c9, 0x57140bdf, 0x30aebcf7,
	0xee930000, 0xd6070000, 0x92c10000, 0x2b9801e0, 0x9451287c, 0x3b6cfb57, 0x45312374, 0x201f6a64, 0x7b280000, 0x57420000, 0xa9e50000, 0x634300a0, 0x9edb442f, 0x6d9995bb, 0x27f83b03, 0xc7ff60f0,
	0x7b280000, 0x57420000, 0xa9e50000, 0x634300a0, 0x9edb442f, 0x6d9995bb, 0x27f83b03, 0xc7ff60f0, 0x95bb0000, 0x81450000, 0x3b240000, 0x48db0140, 0x0a8a6c53, 0x56f56eec, 0x62c91877, 0xe7e00a94
};

__device__ __forceinline__
static void hamsi_block(uint32_t *m, const uint32_t* tp, const uint32_t dm){

	#pragma unroll 16
	for(int i=0;i<16;i++){
		m[ i] ^= dm & tp[ i];
	}
}

#define SBOX(a, b, c, d) { \
	uint32_t t; \
	t = (a); \
	(a) &= (c); \
	(a) ^= (d); \
	(c) ^= (b); \
	(c) ^= (a); \
	(d) |= t; \
	(d) ^= (b); \
	t ^= (c); \
	(b) = (d); \
	(d) |= t; \
	(d) ^= (a); \
	(a) &= (b); \
	t ^= (a); \
	(b) ^= (d); \
	(a) = (c); \
	(c) = (b) ^ t; \
	(b) = (d); \
	(d) = ~t; \
}

#define HAMSI_L(a, b, c, d) { \
		(a) = ROTL32(a, 13); \
		(c) = ROTL32(c, 3); \
		(b) ^= (a) ^ (c); \
		(d) ^= (c) ^ ((a) << 3); \
		(b) = ROTL32(b, 1); \
		(d) = ROTL32(d, 7); \
		(a) = ROTL32(a ^ b ^ d, 5); \
		(c) = ROTL32(c ^ d ^ (b<<7), 22); \
	}

#define ROUND_BIG(rc, alpha) { \
		m[ 0] ^= alpha[ 0]; \
		m[ 1] ^= alpha[ 1] ^ (rc); \
		c[ 0] ^= alpha[ 2]; \
		c[ 1] ^= alpha[ 3]; \
		m[ 2] ^= alpha[ 4]; \
		m[ 3] ^= alpha[ 5]; \
		c[ 2] ^= alpha[ 6]; \
		c[ 3] ^= alpha[ 7]; \
		c[ 4] ^= alpha[ 8]; \
		c[ 5] ^= alpha[ 9]; \
		m[ 4] ^= alpha[10]; \
		m[ 5] ^= alpha[11]; \
		c[ 6] ^= alpha[12]; \
		c[ 7] ^= alpha[13]; \
		m[ 6] ^= alpha[14]; \
		m[ 7] ^= alpha[15]; \
		m[ 8] ^= alpha[16]; \
		m[ 9] ^= alpha[17]; \
		c[ 8] ^= alpha[18]; \
		c[ 9] ^= alpha[19]; \
		m[10] ^= alpha[20]; \
		m[11] ^= alpha[21]; \
		c[10] ^= alpha[22]; \
		c[11] ^= alpha[23]; \
		c[12] ^= alpha[24]; \
		c[13] ^= alpha[25]; \
		m[12] ^= alpha[26]; \
		m[13] ^= alpha[27]; \
		c[14] ^= alpha[28]; \
		c[15] ^= alpha[29]; \
		m[14] ^= alpha[30]; \
		m[15] ^= alpha[31]; \
		SBOX(m[ 0], c[ 4], m[ 8], c[12]); \
		SBOX(m[ 1], c[ 5], m[ 9], c[13]); \
		SBOX(c[ 0], m[ 4], c[ 8], m[12]); \
		SBOX(c[ 1], m[ 5], c[ 9], m[13]); \
		HAMSI_L(m[ 0], c[ 5], c[ 8], m[13]); \
		SBOX(m[ 2], c[ 6], m[10], c[14]); \
		HAMSI_L(m[ 1], m[ 4], c[ 9], c[14]); \
		SBOX(m[ 3], c[ 7], m[11], c[15]); \
		HAMSI_L(c[ 0], m[ 5], m[10], c[15]); \
		SBOX(c[ 2], m[ 6], c[10], m[14]); \
		HAMSI_L(c[ 1], c[ 6], m[11], m[14]); \
		SBOX(c[ 3], m[ 7], c[11], m[15]); \
		HAMSI_L(m[ 2], c[ 7], c[10], m[15]); \
		HAMSI_L(m[ 3], m[ 6], c[11], c[12]); \
		HAMSI_L(c[ 2], m[ 7], m[ 8], c[13]); \
		HAMSI_L(c[ 3], c[ 4], m[ 9], m[12]); \
		HAMSI_L(m[ 0], c[ 0], m[ 3], c[ 3]); \
		HAMSI_L(m[ 8], c[ 9], m[11], c[10]); \
		HAMSI_L(c[ 5], m[ 5], c[ 6], m[ 6]); \
		HAMSI_L(c[13], m[12], c[14], m[15]); \
	}
//-------END OF HAMSI MACROS--------------------------------------------

//------FUGUE MACROS--------------------------------------------------
static __constant__ const uint32_t c_S[16] = {
		0x8807a57e, 0xe616af75, 0xc5d3e4db, 0xac9ab027,
		0xd915f117, 0xb6eecc54, 0x06e8020b, 0x4a92efd1,
		0xaac6e2c9, 0xddb21398, 0xcae65838, 0x437f203f,
		0x25ea78e7, 0x951fddd6, 0xda6ed11d, 0xe13e3567
};

static __device__ uint32_t mixtab0[256] = {
	0x63633297, 0x7c7c6feb, 0x77775ec7, 0x7b7b7af7, 0xf2f2e8e5, 0x6b6b0ab7,	0x6f6f16a7, 0xc5c56d39, 0x303090c0, 0x01010704, 0x67672e87, 0x2b2bd1ac, 0xfefeccd5, 0xd7d71371, 0xabab7c9a,
	0x767659c3, 0xcaca4005, 0x8282a33e, 0xc9c94909, 0x7d7d68ef, 0xfafad0c5,	0x5959947f, 0x4747ce07, 0xf0f0e6ed, 0xadad6e82, 0xd4d41a7d, 0xa2a243be, 0xafaf608a, 0x9c9cf946, 0xa4a451a6,
	0x727245d3, 0xc0c0762d, 0xb7b728ea, 0xfdfdc5d9, 0x9393d47a, 0x2626f298, 0x363682d8, 0x3f3fbdfc, 0xf7f7f3f1, 0xcccc521d, 0x34348cd0, 0xa5a556a2, 0xe5e58db9, 0xf1f1e1e9, 0x71714cdf,
	0xd8d83e4d, 0x313197c4, 0x15156b54, 0x04041c10, 0xc7c76331, 0x2323e98c, 0xc3c37f21, 0x18184860, 0x9696cf6e, 0x05051b14, 0x9a9aeb5e, 0x0707151c, 0x12127e48, 0x8080ad36, 0xe2e298a5,
	0xebeba781, 0x2727f59c, 0xb2b233fe, 0x757550cf, 0x09093f24, 0x8383a43a, 0x2c2cc4b0, 0x1a1a4668, 0x1b1b416c, 0x6e6e11a3, 0x5a5a9d73, 0xa0a04db6, 0x5252a553, 0x3b3ba1ec, 0xd6d61475,
	0xb3b334fa, 0x2929dfa4, 0xe3e39fa1, 0x2f2fcdbc, 0x8484b126, 0x5353a257, 0xd1d10169, 0x00000000, 0xededb599, 0x2020e080, 0xfcfcc2dd, 0xb1b13af2, 0x5b5b9a77, 0x6a6a0db3, 0xcbcb4701,
	0xbebe17ce, 0x3939afe4, 0x4a4aed33, 0x4c4cff2b, 0x5858937b, 0xcfcf5b11, 0xd0d0066d, 0xefefbb91, 0xaaaa7b9e, 0xfbfbd7c1, 0x4343d217, 0x4d4df82f, 0x333399cc, 0x8585b622, 0x4545c00f,
	0xf9f9d9c9, 0x02020e08, 0x7f7f66e7, 0x5050ab5b, 0x3c3cb4f0, 0x9f9ff04a, 0xa8a87596, 0x5151ac5f, 0xa3a344ba, 0x4040db1b, 0x8f8f800a, 0x9292d37e, 0x9d9dfe42, 0x3838a8e0, 0xf5f5fdf9,
	0xbcbc19c6, 0xb6b62fee, 0xdada3045, 0x2121e784, 0x10107040, 0xffffcbd1, 0xf3f3efe1, 0xd2d20865, 0xcdcd5519, 0x0c0c2430, 0x1313794c, 0xececb29d, 0x5f5f8667, 0x9797c86a, 0x4444c70b,
	0x1717655c, 0xc4c46a3d, 0xa7a758aa, 0x7e7e61e3, 0x3d3db3f4, 0x6464278b, 0x5d5d886f, 0x19194f64, 0x737342d7, 0x60603b9b, 0x8181aa32, 0x4f4ff627, 0xdcdc225d, 0x2222ee88, 0x2a2ad6a8,
	0x9090dd76, 0x88889516, 0x4646c903, 0xeeeebc95, 0xb8b805d6, 0x14146c50, 0xdede2c55, 0x5e5e8163, 0x0b0b312c, 0xdbdb3741, 0xe0e096ad, 0x32329ec8, 0x3a3aa6e8, 0x0a0a3628, 0x4949e43f,
	0x06061218, 0x2424fc90, 0x5c5c8f6b, 0xc2c27825, 0xd3d30f61, 0xacac6986, 0x62623593, 0x9191da72, 0x9595c662, 0xe4e48abd, 0x797974ff, 0xe7e783b1, 0xc8c84e0d, 0x373785dc, 0x6d6d18af,
	0x8d8d8e02, 0xd5d51d79, 0x4e4ef123, 0xa9a97292, 0x6c6c1fab, 0x5656b943, 0xf4f4fafd, 0xeaeaa085, 0x6565208f, 0x7a7a7df3, 0xaeae678e, 0x08083820, 0xbaba0bde, 0x787873fb, 0x2525fb94,
	0x2e2ecab8, 0x1c1c5470, 0xa6a65fae, 0xb4b421e6, 0xc6c66435, 0xe8e8ae8d, 0xdddd2559, 0x747457cb, 0x1f1f5d7c, 0x4b4bea37, 0xbdbd1ec2, 0x8b8b9c1a, 0x8a8a9b1e, 0x70704bdb, 0x3e3ebaf8,
	0xb5b526e2, 0x66662983, 0x4848e33b, 0x0303090c, 0xf6f6f4f5, 0x0e0e2a38, 0x61613c9f, 0x35358bd4, 0x5757be47, 0xb9b902d2, 0x8686bf2e, 0xc1c17129, 0x1d1d5374, 0x9e9ef74e, 0xe1e191a9,
	0xf8f8decd, 0x9898e556, 0x11117744, 0x696904bf, 0xd9d93949, 0x8e8e870e, 0x9494c166, 0x9b9bec5a, 0x1e1e5a78, 0x8787b82a, 0xe9e9a989, 0xcece5c15, 0x5555b04f, 0x2828d8a0, 0xdfdf2b51,
	0x8c8c8906, 0xa1a14ab2, 0x89899212, 0x0d0d2334, 0xbfbf10ca, 0xe6e684b5, 0x4242d513, 0x686803bb, 0x4141dc1f, 0x9999e252, 0x2d2dc3b4, 0x0f0f2d3c, 0xb0b03df6, 0x5454b74b, 0xbbbb0cda,
	0x16166258
};


#define mixtab0(x) shared[0][x]
#define mixtab1(x) shared[1][x]
#define mixtab2(x) shared[2][x]
#define mixtab3(x) shared[3][x]

#define TIX4(q, x00, x01, x04, x07, x08, x22, x24, x27, x30) { \
		x22 ^= x00; \
		x00 = (q); \
		x08 ^= (q); \
		x01 ^= x24; \
		x04 ^= x27; \
		x07 ^= x30; \
	}

#define CMIX36(x00, x01, x02, x04, x05, x06, x18, x19, x20) { \
		x00 ^= x04; \
		x01 ^= x05; \
		x02 ^= x06; \
		x18 ^= x04; \
		x19 ^= x05; \
		x20 ^= x06; \
	}

__device__ __forceinline__
static void SMIX_LDG(const uint32_t shared[4][256], uint32_t &x0,uint32_t &x1,uint32_t &x2,uint32_t &x3){
	uint32_t c0 = __ldg(&mixtab0[__byte_perm(x0,0,0x4443)]);
	uint32_t r1 = mixtab1(__byte_perm(x0,0,0x4442));
	uint32_t r2 = mixtab2(__byte_perm(x0,0,0x4441));
	uint32_t r3 = mixtab3(__byte_perm(x0,0,0x4440));
	c0 = c0 ^ r1 ^ r2 ^ r3;
	uint32_t r0 = mixtab0(__byte_perm(x1,0,0x4443));
	uint32_t c1 = r0 ^ mixtab1(__byte_perm(x1,0,0x4442));
	uint32_t tmp = mixtab2(__byte_perm(x1,0,0x4441));
	c1 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x1,0,0x4440));
	c1 ^= tmp;
	r3 ^= tmp;
	uint32_t c2 = __ldg(&mixtab0[__byte_perm(x2,0,0x4443)]);
	r0 ^= c2;
	tmp = mixtab1(__byte_perm(x2,0,0x4442));
	c2 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x2,0,0x4441));
	c2 ^= tmp;
	tmp = mixtab3(__byte_perm(x2,0,0x4440));
	c2 ^= tmp;
	r3 ^= tmp;
	uint32_t c3 = __ldg(&mixtab0[__byte_perm(x3,0,0x4443)]);
	r0 ^= c3;
	tmp = mixtab1(__byte_perm(x3,0,0x4442));
	c3 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x3,0,0x4441));
	c3 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x3,0,0x4440));
	c3 ^= tmp;
	x0 = ((c0 ^ (r0 << 0)) & 0xFF000000) | ((c1 ^ (r1 << 0)) & 0x00FF0000) | ((c2 ^ (r2 << 0)) & 0x0000FF00) | ((c3 ^ (r3 << 0)) & 0x000000FF);
	x1 = ((c1 ^ (r0 << 8)) & 0xFF000000) | ((c2 ^ (r1 << 8)) & 0x00FF0000) | ((c3 ^ (r2 << 8)) & 0x0000FF00) | ((c0 ^ (r3 >>24)) & 0x000000FF);
	x2 = ((c2 ^ (r0 <<16)) & 0xFF000000) | ((c3 ^ (r1 <<16)) & 0x00FF0000) | ((c0 ^ (r2 >>16)) & 0x0000FF00) | ((c1 ^ (r3 >>16)) & 0x000000FF);
	x3 = ((c3 ^ (r0 <<24)) & 0xFF000000) | ((c0 ^ (r1 >> 8)) & 0x00FF0000) | ((c1 ^ (r2 >> 8)) & 0x0000FF00) | ((c2 ^ (r3 >> 8)) & 0x000000FF);
}

__device__ __forceinline__
static void SMIX(const uint32_t shared[4][256], uint32_t &x0,uint32_t &x1,uint32_t &x2,uint32_t &x3){
	uint32_t c0 = mixtab0(__byte_perm(x0,0,0x4443));
	uint32_t r1 = mixtab1(__byte_perm(x0,0,0x4442));
	uint32_t r2 = mixtab2(__byte_perm(x0,0,0x4441));
	uint32_t r3 = mixtab3(__byte_perm(x0,0,0x4440));
	c0 = c0 ^ r1 ^ r2 ^ r3;
	uint32_t r0 = mixtab0(__byte_perm(x1,0,0x4443));
	uint32_t c1 = r0 ^ mixtab1(__byte_perm(x1,0,0x4442));
	uint32_t tmp = mixtab2(__byte_perm(x1,0,0x4441));
	c1 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x1,0,0x4440));
	c1 ^= tmp;
	r3 ^= tmp;
	uint32_t c2 = mixtab0(__byte_perm(x2,0,0x4443));
	r0 ^= c2;
	tmp = mixtab1(__byte_perm(x2,0,0x4442));
	c2 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x2,0,0x4441));
	c2 ^= tmp;
	tmp = mixtab3(__byte_perm(x2,0,0x4440));
	c2 ^= tmp;
	r3 ^= tmp;
	uint32_t c3 = mixtab0(__byte_perm(x3,0,0x4443));
	r0 ^= c3;
	tmp = mixtab1(__byte_perm(x3,0,0x4442));
	c3 ^= tmp;
	r1 ^= tmp;
	tmp = mixtab2(__byte_perm(x3,0,0x4441));
	c3 ^= tmp;
	r2 ^= tmp;
	tmp = mixtab3(__byte_perm(x3,0,0x4440));
	c3 ^= tmp;
	x0 = ((c0 ^ (r0 << 0)) & 0xFF000000) | ((c1 ^ (r1 << 0)) & 0x00FF0000) | ((c2 ^ (r2 << 0)) & 0x0000FF00) | ((c3 ^ (r3 << 0)) & 0x000000FF);
	x1 = ((c1 ^ (r0 << 8)) & 0xFF000000) | ((c2 ^ (r1 << 8)) & 0x00FF0000) | ((c3 ^ (r2 << 8)) & 0x0000FF00) | ((c0 ^ (r3 >>24)) & 0x000000FF);
	x2 = ((c2 ^ (r0 <<16)) & 0xFF000000) | ((c3 ^ (r1 <<16)) & 0x00FF0000) | ((c0 ^ (r2 >>16)) & 0x0000FF00) | ((c1 ^ (r3 >>16)) & 0x000000FF);
	x3 = ((c3 ^ (r0 <<24)) & 0xFF000000) | ((c0 ^ (r1 >> 8)) & 0x00FF0000) | ((c1 ^ (r2 >> 8)) & 0x0000FF00) | ((c2 ^ (r3 >> 8)) & 0x000000FF);
}

#define mROR3 { \
	B[ 6] = S[33], B[ 7] = S[34], B[ 8] = S[35]; \
	S[35] = S[32]; S[34] = S[31]; S[33] = S[30]; S[32] = S[29]; S[31] = S[28]; S[30] = S[27]; S[29] = S[26]; S[28] = S[25]; S[27] = S[24]; \
	S[26] = S[23]; S[25] = S[22]; S[24] = S[21]; S[23] = S[20]; S[22] = S[19]; S[21] = S[18]; S[20] = S[17]; S[19] = S[16]; S[18] = S[15]; \
	S[17] = S[14]; S[16] = S[13]; S[15] = S[12]; S[14] = S[11]; S[13] = S[10]; S[12] = S[ 9]; S[11] = S[ 8]; S[10] = S[ 7]; S[ 9] = S[ 6]; \
	S[ 8] = S[ 5]; S[ 7] = S[ 4]; S[ 6] = S[ 3]; S[ 5] = S[ 2]; S[ 4] = S[ 1]; S[ 3] = S[ 0]; S[ 2] = B[ 8]; S[ 1] = B[ 7]; S[ 0] = B[ 6]; \
	}

#define mROR8 { \
	B[ 1] = S[28]; B[ 2] = S[29]; B[ 3] = S[30]; B[ 4] = S[31]; B[ 5] = S[32]; B[ 6] = S[33]; B[ 7] = S[34]; B[ 8] = S[35]; \
	S[35] = S[27]; S[34] = S[26]; S[33] = S[25]; S[32] = S[24]; S[31] = S[23]; S[30] = S[22]; S[29] = S[21]; S[28] = S[20]; S[27] = S[19]; \
	S[26] = S[18]; S[25] = S[17]; S[24] = S[16]; S[23] = S[15]; S[22] = S[14]; S[21] = S[13]; S[20] = S[12]; S[19] = S[11]; S[18] = S[10]; \
	S[17] = S[ 9]; S[16] = S[ 8]; S[15] = S[ 7]; S[14] = S[ 6]; S[13] = S[ 5]; S[12] = S[ 4]; S[11] = S[ 3]; S[10] = S[ 2]; S[ 9] = S[ 1]; \
	S[ 8] = S[ 0]; S[ 7] = B[ 8]; S[ 6] = B[ 7]; S[ 5] = B[ 6]; S[ 4] = B[ 5]; S[ 3] = B[ 4]; S[ 2] = B[ 3]; S[ 1] = B[ 2]; S[ 0] = B[ 1]; \
	}

#define mROR9 { \
	B[ 0] = S[27]; B[ 1] = S[28]; B[ 2] = S[29]; B[ 3] = S[30]; B[ 4] = S[31]; B[ 5] = S[32]; B[ 6] = S[33]; B[ 7] = S[34]; B[ 8] = S[35]; \
	S[35] = S[26]; S[34] = S[25]; S[33] = S[24]; S[32] = S[23]; S[31] = S[22]; S[30] = S[21]; S[29] = S[20]; S[28] = S[19]; S[27] = S[18]; \
	S[26] = S[17]; S[25] = S[16]; S[24] = S[15]; S[23] = S[14]; S[22] = S[13]; S[21] = S[12]; S[20] = S[11]; S[19] = S[10]; S[18] = S[ 9]; \
	S[17] = S[ 8]; S[16] = S[ 7]; S[15] = S[ 6]; S[14] = S[ 5]; S[13] = S[ 4]; S[12] = S[ 3]; S[11] = S[ 2]; S[10] = S[ 1]; S[ 9] = S[ 0]; \
	S[ 8] = B[ 8]; S[ 7] = B[ 7]; S[ 6] = B[ 6]; S[ 5] = B[ 5]; S[ 4] = B[ 4]; S[ 3] = B[ 3]; S[ 2] = B[ 2]; S[ 1] = B[ 1]; S[ 0] = B[ 0]; \
	}

#define FUGUE512_3(x, y, z) {  \
        TIX4(x, S[ 0], S[ 1], S[ 4], S[ 7], S[ 8], S[22], S[24], S[27], S[30]); \
        CMIX36(S[33], S[34], S[35], S[ 1], S[ 2], S[ 3], S[15], S[16], S[17]); \
        SMIX(shared, S[33], S[34], S[35], S[ 0]); \
        CMIX36(S[30], S[31], S[32], S[34], S[35], S[ 0], S[12], S[13], S[14]); \
        SMIX(shared, S[30], S[31], S[32], S[33]); \
        CMIX36(S[27], S[28], S[29], S[31], S[32], S[33], S[ 9], S[10], S[11]); \
        SMIX(shared, S[27], S[28], S[29], S[30]); \
        CMIX36(S[24], S[25], S[26], S[28], S[29], S[30], S[ 6], S[ 7], S[ 8]); \
        SMIX_LDG(shared, S[24], S[25], S[26], S[27]); \
        \
        TIX4(y, S[24], S[25], S[28], S[31], S[32], S[10], S[12], S[15], S[18]); \
        CMIX36(S[21], S[22], S[23], S[25], S[26], S[27], S[ 3], S[ 4], S[ 5]); \
        SMIX(shared, S[21], S[22], S[23], S[24]); \
        CMIX36(S[18], S[19], S[20], S[22], S[23], S[24], S[ 0], S[ 1], S[ 2]); \
        SMIX_LDG(shared, S[18], S[19], S[20], S[21]); \
        CMIX36(S[15], S[16], S[17], S[19], S[20], S[21], S[33], S[34], S[35]); \
        SMIX(shared, S[15], S[16], S[17], S[18]); \
        CMIX36(S[12], S[13], S[14], S[16], S[17], S[18], S[30], S[31], S[32]); \
        SMIX_LDG(shared, S[12], S[13], S[14], S[15]); \
        \
        TIX4(z, S[12], S[13], S[16], S[19], S[20], S[34], S[ 0], S[ 3], S[ 6]); \
        CMIX36(S[ 9], S[10], S[11], S[13], S[14], S[15], S[27], S[28], S[29]); \
        SMIX(shared, S[ 9], S[10], S[11], S[12]); \
        CMIX36(S[ 6], S[ 7], S[ 8], S[10], S[11], S[12], S[24], S[25], S[26]); \
        SMIX_LDG(shared, S[ 6], S[ 7], S[ 8], S[ 9]); \
        CMIX36(S[ 3], S[ 4], S[ 5], S[ 7], S[ 8], S[ 9], S[21], S[22], S[23]); \
        SMIX_LDG(shared, S[ 3], S[ 4], S[ 5], S[ 6]); \
        CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]); \
        SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]); \
	}

//------END OF FUGUE MACROS-------------------------------------------

__global__ 
#if __CUDA_ARCH__ > 500
__launch_bounds__(384,2)
#else
__launch_bounds__(384)
#endif
void x13_hamsi_fugue512_gpu_hash_64(uint32_t threads, uint32_t *g_hash){

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	__shared__ uint32_t shared[4][256];

	if(threadIdx.x<256){
		const uint32_t tmp = mixtab0[threadIdx.x];
		shared[0][threadIdx.x] = tmp;
		shared[1][threadIdx.x] = ROR8(tmp);
		shared[2][threadIdx.x] = ROL16(tmp);
		shared[3][threadIdx.x] = ROL8(tmp);
	}	
	if (thread < threads){

		uint32_t *Hash = &g_hash[thread<<4];
		uint8_t h1[64];
		*(uint2x4*)&h1[ 0] = __ldg4((uint2x4*)&Hash[0]);
		*(uint2x4*)&h1[32] = __ldg4((uint2x4*)&Hash[8]);

		__syncthreads();
		
		uint32_t c[16], h[16], m[16];
		#pragma unroll 16
		for(int i=0;i<16;i++){
			c[i]=c_c[i];
			h[i]=c_c[i];
		}

		const uint32_t *tp;
		uint32_t dm;

		for(int i = 0; i < 64; i += 8) {
			tp = &d_T512[0];

			dm = -(h1[i] & 1);
			m[ 0] = dm & tp[ 0]; m[ 1] = dm & tp[ 1];
			m[ 2] = dm & tp[ 2]; m[ 3] = dm & tp[ 3];
			m[ 4] = dm & tp[ 4]; m[ 5] = dm & tp[ 5];
			m[ 6] = dm & tp[ 6]; m[ 7] = dm & tp[ 7];
			m[ 8] = dm & tp[ 8]; m[ 9] = dm & tp[ 9];
			m[10] = dm & tp[10]; m[11] = dm & tp[11];
			m[12] = dm & tp[12]; m[13] = dm & tp[13];
			m[14] = dm & tp[14]; m[15] = dm & tp[15];
			tp += 16;
			#pragma unroll 7
			for (int v = 1; v < 8; v ++) {
				dm = -((h1[i]>>v) & 1);
				hamsi_block(m,tp,dm);
				tp += 16;
			}
			#pragma unroll
			for (int u = 1; u < 8; u ++) {
				#pragma unroll 8
				for (int v = 0; v < 8; v ++) {
					dm = -((h1[i+u]>>v) & 1);
					hamsi_block(m,tp,dm);
					tp += 16;
				}
			}

			#pragma unroll 6
			for( int r = 0; r < 6; r++ ) {
				ROUND_BIG(r, d_alpha_n);
			}
			/* order is (no more) important */
			h[ 0] ^= m[ 0]; h[ 1] ^= m[ 1]; h[ 2] ^= c[ 0]; h[ 3] ^= c[ 1];
			h[ 4] ^= m[ 2]; h[ 5] ^= m[ 3]; h[ 6] ^= c[ 2]; h[ 7] ^= c[ 3];
			h[ 8] ^= m[ 8]; h[ 9] ^= m[ 9]; h[10] ^= c[ 8]; h[11] ^= c[ 9];
			h[12] ^= m[10]; h[13] ^= m[11]; h[14] ^= c[10]; h[15] ^= c[11];

			#pragma unroll 16
			for(int i=0;i<16;i++)
				c[i]=h[i];
		}

		*(uint2x4*)&m[ 0] = *(uint2x4*)&d_T512[112];
		*(uint2x4*)&m[ 8] = *(uint2x4*)&d_T512[120];

		#pragma unroll 6
		for( int r = 0; r < 6; r++ ) {
			ROUND_BIG(r, d_alpha_n);
		}

		/* order is (no more) important */
		h[ 0] ^= m[ 0]; h[ 1] ^= m[ 1]; h[ 2] ^= c[ 0]; h[ 3] ^= c[ 1];
		h[ 4] ^= m[ 2]; h[ 5] ^= m[ 3]; h[ 6] ^= c[ 2]; h[ 7] ^= c[ 3];
		h[ 8] ^= m[ 8]; h[ 9] ^= m[ 9]; h[10] ^= c[ 8]; h[11] ^= c[ 9];
		h[12] ^= m[10]; h[13] ^= m[11]; h[14] ^= c[10]; h[15] ^= c[11];
			
		#pragma unroll 16
		for(int i=0;i<16;i++)
			c[i]=h[i];

		*(uint2x4*)&m[ 0] = *(uint2x4*)&d_T512[784];
		*(uint2x4*)&m[ 8] = *(uint2x4*)&d_T512[792];

//		#pragma unroll 12
		for( int r = 0; r < 12; r+=6 ){
			ROUND_BIG(r+0, d_alpha_f);
			ROUND_BIG(r+1, d_alpha_f);
			ROUND_BIG(r+2, d_alpha_f);
			ROUND_BIG(r+3, d_alpha_f);
			ROUND_BIG(r+4, d_alpha_f);
			ROUND_BIG(r+5, d_alpha_f);
		}
		
		/* order is (no more) important */
		h[ 0] ^= m[ 0]; h[ 1] ^= m[ 1]; h[ 2] ^= c[ 0]; h[ 3] ^= c[ 1];
		h[ 4] ^= m[ 2]; h[ 5] ^= m[ 3]; h[ 6] ^= c[ 2]; h[ 7] ^= c[ 3];
		h[ 8] ^= m[ 8]; h[ 9] ^= m[ 9]; h[10] ^= c[ 8]; h[11] ^= c[ 9];
		h[12] ^= m[10]; h[13] ^= m[11]; h[14] ^= c[10]; h[15] ^= c[11];
		
//		*(uint2x4*)&Hash[ 0] = *(uint2x4*)&h[ 0];
//		*(uint2x4*)&Hash[ 8] = *(uint2x4*)&h[ 8];
		uint32_t S[36];
		uint32_t B[ 9];
		
		S[ 0] = S[ 1] = S[ 2] = S[ 3] = S[ 4] = S[ 5] = S[ 6] = S[ 7] = S[ 8] = S[ 9] = S[10] = S[11] = S[12] = S[13] = S[14] = S[15] = S[16] = S[17] = S[18] = S[19] = 0;
		*(uint2x4*)&S[20] = *(uint2x4*)&c_S[ 0];
		#pragma unroll 8
		for(int i=0;i<8;i++){
			S[28+i] = c_S[i+8];
		}

		FUGUE512_3(h[0x0], h[0x1], h[0x2]);
		FUGUE512_3(h[0x3], h[0x4], h[0x5]);
		FUGUE512_3(h[0x6], h[0x7], h[0x8]);
		FUGUE512_3(h[0x9], h[0xA], h[0xB]);
		FUGUE512_3(h[0xC], h[0xD], h[0xE]);
		FUGUE512_3(h[0xF], 0U, 512U);

		for (uint32_t i = 0; i < 32; i+=2){
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		#pragma unroll 11
		for (uint32_t i = 0; i < 13; i ++) {
			S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[28] ^= S[ 0];
			mROR8;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];

		S[ 0] = cuda_swab32(S[ 1]);	S[ 1] = cuda_swab32(S[ 2]);	S[ 2] = cuda_swab32(S[ 3]);	S[ 3] = cuda_swab32(S[ 4]);
		S[ 4] = cuda_swab32(S[ 9]);	S[ 5] = cuda_swab32(S[10]);	S[ 6] = cuda_swab32(S[11]);	S[ 7] = cuda_swab32(S[12]);
		S[ 8] = cuda_swab32(S[18]);	S[ 9] = cuda_swab32(S[19]);	S[10] = cuda_swab32(S[20]);	S[11] = cuda_swab32(S[21]);
		S[12] = cuda_swab32(S[27]);	S[13] = cuda_swab32(S[28]);	S[14] = cuda_swab32(S[29]);	S[15] = cuda_swab32(S[30]);
		
		*(uint2x4*)&Hash[ 0] = *(uint2x4*)&S[ 0];
		*(uint2x4*)&Hash[ 8] = *(uint2x4*)&S[ 8];
	}
}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(384,2)
#else
__launch_bounds__(384)
#endif
void x13_hamsi_fugue512_gpu_hash_64_final(uint32_t threads, uint32_t *g_hash,uint32_t* resNonce, const uint64_t target){

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	__shared__ uint32_t shared[4][256];

	if(threadIdx.x<256){
		const uint32_t tmp = mixtab0[threadIdx.x];
		shared[0][threadIdx.x] = tmp;
		shared[1][threadIdx.x] = ROR8(tmp);
		shared[2][threadIdx.x] = ROL16(tmp);
		shared[3][threadIdx.x] = ROL8(tmp);	
	}

	if (thread < threads){
		uint32_t *Hash = &g_hash[thread<<4];
		uint8_t h1[64];
		*(uint2x4*)&h1[ 0] = __ldg4((uint2x4*)&Hash[0]);
		*(uint2x4*)&h1[32] = __ldg4((uint2x4*)&Hash[8]);

		__syncthreads();

		uint32_t c[16], h[16], m[16];
		#pragma unroll 16
		for(int i=0;i<16;i++){
			c[i]=c_c[i];
			h[i]=c_c[i];
		}

		const uint32_t *tp;
		uint32_t dm;

		for(int i = 0; i < 64; i += 8) {
			tp = &d_T512[0];

			dm = -(h1[i] & 1);
			m[ 0] = dm & tp[ 0]; m[ 1] = dm & tp[ 1];
			m[ 2] = dm & tp[ 2]; m[ 3] = dm & tp[ 3];
			m[ 4] = dm & tp[ 4]; m[ 5] = dm & tp[ 5];
			m[ 6] = dm & tp[ 6]; m[ 7] = dm & tp[ 7];
			m[ 8] = dm & tp[ 8]; m[ 9] = dm & tp[ 9];
			m[10] = dm & tp[10]; m[11] = dm & tp[11];
			m[12] = dm & tp[12]; m[13] = dm & tp[13];
			m[14] = dm & tp[14]; m[15] = dm & tp[15];
			tp += 16;
			#pragma unroll 7
			for (int v = 1; v < 8; v ++) {
				dm = -((h1[i]>>v) & 1);
				hamsi_block(m,tp,dm);
				tp += 16;
			}
			#pragma unroll
			for (int u = 1; u < 8; u ++) {
				#pragma unroll 8
				for (int v = 0; v < 8; v ++) {
					dm = -((h1[i+u]>>v) & 1);
					hamsi_block(m,tp,dm);
					tp += 16;
				}
			}

			#pragma unroll 6
			for( int r = 0; r < 6; r++ ) {
				ROUND_BIG(r, d_alpha_n);
			}
			/* order is (no more) important */
			h[ 0] ^= m[ 0]; h[ 1] ^= m[ 1]; h[ 2] ^= c[ 0]; h[ 3] ^= c[ 1];
			h[ 4] ^= m[ 2]; h[ 5] ^= m[ 3]; h[ 6] ^= c[ 2]; h[ 7] ^= c[ 3];
			h[ 8] ^= m[ 8]; h[ 9] ^= m[ 9]; h[10] ^= c[ 8]; h[11] ^= c[ 9];
			h[12] ^= m[10]; h[13] ^= m[11]; h[14] ^= c[10]; h[15] ^= c[11];

			#pragma unroll 16
			for(int i=0;i<16;i++){
				c[i]=h[i];
			}
		}

		*(uint2x4*)&m[ 0] = *(uint2x4*)&d_T512[112];
		*(uint2x4*)&m[ 8] = *(uint2x4*)&d_T512[120];

		#pragma unroll 6
		for( int r = 0; r < 6; r++ ) {
			ROUND_BIG(r, d_alpha_n);
		}

		/* order is (no more) important */
		h[ 0] ^= m[ 0]; h[ 1] ^= m[ 1]; h[ 2] ^= c[ 0]; h[ 3] ^= c[ 1];
		h[ 4] ^= m[ 2]; h[ 5] ^= m[ 3]; h[ 6] ^= c[ 2]; h[ 7] ^= c[ 3];
		h[ 8] ^= m[ 8]; h[ 9] ^= m[ 9]; h[10] ^= c[ 8]; h[11] ^= c[ 9];
		h[12] ^= m[10]; h[13] ^= m[11]; h[14] ^= c[10]; h[15] ^= c[11];
			
		#pragma unroll 16
		for(int i=0;i<16;i++){
			c[i]=h[i];
		}

		*(uint2x4*)&m[ 0] = *(uint2x4*)&d_T512[784];
		*(uint2x4*)&m[ 8] = *(uint2x4*)&d_T512[792];

//		#pragma unroll 12
		for( int r = 0; r < 12; r+=6 ){
			ROUND_BIG(r+0, d_alpha_f);
			ROUND_BIG(r+1, d_alpha_f);
			ROUND_BIG(r+2, d_alpha_f);
			ROUND_BIG(r+3, d_alpha_f);
			ROUND_BIG(r+4, d_alpha_f);
			ROUND_BIG(r+5, d_alpha_f);
		}
		
		/* order is (no more) important */
		h[ 0] ^= m[ 0]; h[ 1] ^= m[ 1]; h[ 2] ^= c[ 0]; h[ 3] ^= c[ 1];
		h[ 4] ^= m[ 2]; h[ 5] ^= m[ 3]; h[ 6] ^= c[ 2]; h[ 7] ^= c[ 3];
		h[ 8] ^= m[ 8]; h[ 9] ^= m[ 9]; h[10] ^= c[ 8]; h[11] ^= c[ 9];
		h[12] ^= m[10]; h[13] ^= m[11]; h[14] ^= c[10]; h[15] ^= c[11];
		
//		*(uint2x4*)&Hash[ 0] = *(uint2x4*)&h[ 0];
//		*(uint2x4*)&Hash[ 8] = *(uint2x4*)&h[ 8];
		uint32_t S[36];
		uint32_t B[ 9];
		
		S[ 0] = S[ 1] = S[ 2] = S[ 3] = S[ 4] = S[ 5] = S[ 6] = S[ 7] = S[ 8] = S[ 9] = S[10] = S[11] = S[12] = S[13] = S[14] = S[15] = S[16] = S[17] = S[18] = S[19] = 0;
		*(uint2x4*)&S[20] = *(uint2x4*)&c_S[ 0];
		#pragma unroll 8
		for(int i=0;i<8;i++){
			S[28+i] = c_S[i+8];
		}

		FUGUE512_3(h[0x0], h[0x1], h[0x2]);
		FUGUE512_3(h[0x3], h[0x4], h[0x5]);
		FUGUE512_3(h[0x6], h[0x7], h[0x8]);
		FUGUE512_3(h[0x9], h[0xA], h[0xB]);
		FUGUE512_3(h[0xC], h[0xD], h[0xE]);
		FUGUE512_3(h[0xF], 0U, 512U);

		for (uint32_t i = 0; i < 32; i+=2){
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			mROR3;
			CMIX36(S[ 0], S[ 1], S[ 2], S[ 4], S[ 5], S[ 6], S[18], S[19], S[20]);
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		#pragma unroll 10
		for (int i = 0; i < 12; i++) {
			S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[27] ^= S[ 0];
			mROR9;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
			S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[28] ^= S[ 0];
			mROR8;
			SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		}
		S[ 4] ^= S[ 0];	S[ 9] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
		mROR9;
		SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[18] ^= S[ 0];	S[27] ^= S[ 0];
		mROR9;
		SMIX(shared, S[ 0], S[ 1], S[ 2], S[ 3]);
		S[ 4] ^= S[ 0];	S[10] ^= S[ 0];	S[19] ^= S[ 0];	S[27] ^= S[ 0];
		mROR9;
		SMIX_LDG(shared, S[ 0], S[ 1], S[ 2], S[ 3]);

		S[ 3] = cuda_swab32(S[3]);	S[ 4] = cuda_swab32(S[4]^S[ 0]);
		
		const uint64_t check = *(uint64_t*)&S[ 3];
		if(check <= target){
			uint32_t tmp = atomicExch(&resNonce[0], thread);
			if (tmp != UINT32_MAX)
				resNonce[1] = tmp;		
		}
	}
}

__host__
void x13_hamsi_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 384;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x13_hamsi_fugue512_gpu_hash_64<<<grid, block>>>(threads, d_hash);

}

__host__
void x13_hamsi_fugue512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target){

	const uint32_t threadsperblock = 384;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x13_hamsi_fugue512_gpu_hash_64_final<<<grid, block>>>(threads, d_hash,d_resNonce,target);
}
