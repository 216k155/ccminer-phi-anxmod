#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"

#include "quark/cuda_quark.h"


static uint32_t *d_hash[MAX_GPUS];
#define NBN 2

/* 8 adapters max */
static uint32_t	*d_resNonce[MAX_GPUS];
static uint32_t	*h_resNonce[MAX_GPUS];

// Original nist5hash Funktion aus einem miner Quelltext
extern "C" void nist5hash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    uint8_t hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_nist5(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done){

	int dev_id = device_map[thr_id];

	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << 21); // 256*256*16
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00FF;

	const uint64_t highTarget = *(uint64_t*)&ptarget[6];
	
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		
		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		h_resNonce[thr_id] = (uint32_t*) malloc(NBN * sizeof(uint32_t));
		if(h_resNonce[thr_id] == NULL){
			gpulog(LOG_ERR,thr_id,"Host memory allocation failed");
			exit(EXIT_FAILURE);
		}
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));				
	int rc = 0;
	do {
		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		quark_groestl512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);
		quark_jh512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);
		quark_keccak_skein512_cpu_hash_64_final(thr_id, throughput, NULL, d_hash[thr_id],d_resNonce[thr_id],highTarget);
		hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);

		if (h_resNonce[thr_id][0] != UINT32_MAX){
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], startNounce+h_resNonce[thr_id][0]);
			nist5hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				rc = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				work_set_target_ratio(work, vhash64);
				pdata[19] =startNounce + h_resNonce[thr_id][0];
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					be32enc(&endiandata[19], startNounce+h_resNonce[thr_id][1]);
					nist5hash(vhash64, endiandata);
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0])
						work_set_target_ratio(work, vhash64);
					pdata[21] = startNounce+h_resNonce[thr_id][1];
					rc=2;
				}
				return rc;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[thr_id][0]);
				hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
			}
		}
		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && (uint64_t)max_nonce > (uint64_t)throughput + (uint64_t)pdata[19]);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// ressources cleanup
extern "C" void free_nist5(int thr_id){
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	free(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);
	hipFree(d_hash[thr_id]);
	
//	blake512_cpu_free(thr_id);
	
	init[thr_id] = false;

	hipDeviceSynchronize();
}
