#include "hip/hip_runtime.h"
/*
//	Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice
	Based upon Christians, Tanguy Pruvot's and SP's work
		
	Provos Alexis - 2016
*/

#include "cuda_vectors.h"
#include "cuda_helper.h"

#define TPB52 512
#define TPB50 512
#define THF 4

#include "quark/groestl_functions_quad.h"
#include "quark/groestl_transf_quad.h"

__constant__ const uint32_t msg[2][4] = {
						{0x00000080,0,0,0},
						{0,0,0,0x01000000}
					};

#if __CUDA_ARCH__ > 500
__global__ __launch_bounds__(TPB52, 2)
#else
__global__ __launch_bounds__(TPB50, 2)
#endif
void quark_groestl512_gpu_hash_64_quad(uint32_t threads, uint32_t* g_hash, uint32_t* g_nonceVector){
	uint32_t msgBitsliced[8];
	uint32_t state[8];
	uint32_t output[16];
	// durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	if (thread < threads){
	        // GROESTL
		const uint32_t hashPosition = (g_nonceVector == NULL) ? thread : __ldg(&g_nonceVector[thread]);

	        uint32_t *inpHash = &g_hash[hashPosition<<4];

	        const uint32_t thr = threadIdx.x & (THF-1);

		uint32_t message[8] = {
			#if __CUDA_ARCH__ > 500
			__ldg(&inpHash[thr]), __ldg(&inpHash[(THF)+thr]), __ldg(&inpHash[(2 * THF) + thr]), __ldg(&inpHash[(3 * THF) + thr]),msg[0][thr], 0, 0, msg[1][thr]
			#else
			inpHash[thr], inpHash[(THF)+thr], inpHash[(2 * THF) + thr], inpHash[(3 * THF) + thr], msg[0][thr], 0, 0, msg[1][thr]
			#endif
		};

		to_bitslice_quad(message, msgBitsliced);

	        groestl512_progressMessage_quad(state, msgBitsliced,thr);

		from_bitslice_quad52(state, output);

#if __CUDA_ARCH__ <= 500
		output[0] = __byte_perm(output[0], __shfl(output[0], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[2] = __byte_perm(output[2], __shfl(output[2], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[4] = __byte_perm(output[4], __shfl(output[4], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[6] = __byte_perm(output[6], __shfl(output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[8] = __byte_perm(output[8], __shfl(output[8], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[10] = __byte_perm(output[10], __shfl(output[10], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[12] = __byte_perm(output[12], __shfl(output[12], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[14] = __byte_perm(output[14], __shfl(output[14], (threadIdx.x + 1) & 3, 4), 0x7632);
	
		if (thr == 0 || thr == 2){
			output[0 + 1] = __shfl(output[0], (threadIdx.x + 2) & 3, 4);
			output[2 + 1] = __shfl(output[2], (threadIdx.x + 2) & 3, 4);
			output[4 + 1] = __shfl(output[4], (threadIdx.x + 2) & 3, 4);
			output[6 + 1] = __shfl(output[6], (threadIdx.x + 2) & 3, 4);
			output[8 + 1] = __shfl(output[8], (threadIdx.x + 2) & 3, 4);
			output[10 + 1] = __shfl(output[10], (threadIdx.x + 2) & 3, 4);
			output[12 + 1] = __shfl(output[12], (threadIdx.x + 2) & 3, 4);
			output[14 + 1] = __shfl(output[14], (threadIdx.x + 2) & 3, 4);		
			if(thr==0){
				*(uint2x4*)&inpHash[0] = *(uint2x4*)&output[0];
				*(uint2x4*)&inpHash[8] = *(uint2x4*)&output[8];
			}
		}
#else
		output[0] = __byte_perm(output[0], __shfl(output[0], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[0 + 1] = __shfl(output[0], (threadIdx.x + 2) & 3, 4);

		output[2] = __byte_perm(output[2], __shfl(output[2], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[2 + 1] = __shfl(output[2], (threadIdx.x + 2) & 3, 4);
		
		output[4] = __byte_perm(output[4], __shfl(output[4], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[4 + 1] = __shfl(output[4], (threadIdx.x + 2) & 3, 4);
		
		output[6] = __byte_perm(output[6], __shfl(output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[6 + 1] = __shfl(output[6], (threadIdx.x + 2) & 3, 4);
		
		output[8] = __byte_perm(output[8], __shfl(output[8], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[8 + 1] = __shfl(output[8], (threadIdx.x + 2) & 3, 4);

		output[10] = __byte_perm(output[10], __shfl(output[10], (threadIdx.x + 1) & 3, 4), 0x7610);
		output[10 + 1] = __shfl(output[10], (threadIdx.x + 2) & 3, 4);
		
		output[12] = __byte_perm(output[12], __shfl(output[12], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[12 + 1] = __shfl(output[12], (threadIdx.x + 2) & 3, 4);
		
		output[14] = __byte_perm(output[14], __shfl(output[14], (threadIdx.x + 1) & 3, 4), 0x7632);
		output[14 + 1] = __shfl(output[14], (threadIdx.x + 2) & 3, 4);

		if(thr==0){
			*(uint2x4*)&inpHash[0] = *(uint2x4*)&output[0];
			*(uint2x4*)&inpHash[8] = *(uint2x4*)&output[8];
		}
#endif
	}
}

__host__
void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_nonceVector, uint32_t *d_hash){

	const int dev_id = device_map[thr_id];
	// Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
	// mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
	// berechne wie viele Thread Blocks wir brauchen
	uint32_t tpb = (device_sm[dev_id] <= 500) ? TPB50 : TPB52;

	dim3 grid((THF*threads + tpb-1)/tpb);
	dim3 block(tpb);
	quark_groestl512_gpu_hash_64_quad<<<grid, block>>>(threads, d_hash, d_nonceVector);
}
