#include "hip/hip_runtime.h"
// Auf Myriadcoin spezialisierte Version von Groestl inkl. Bitslice
// Based on Tanguy Pruvot's repo
// Provos Alexis - 2016

#include "cuda_helper.h"
#include "miner.h"

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#define __funnelshift_r(x,y,n) (x >> n)
#define atomicExch(p,x) x
#endif

// 64 Registers Variant for Compute 3.0
#include "quark/groestl_functions_quad.h"
#include "quark/groestl_transf_quad.h"

// globaler Speicher für alle HeftyHashes aller Threads
static uint32_t *d_outputHashes[MAX_GPUS];

__constant__ uint32_t _ALIGN(8) c_input[32];
// muss expandiert werden
__constant__ const uint32_t sha256_constantTable[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5, 0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

__constant__ const uint32_t sha256_constantTable2[64] = {
	0xC28A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5, 0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF374, 
	0x649B69C1, 0xF0FE4786, 0x0FE1EDC6, 0x240CF254, 0x4FE9346F, 0x6CC984BE, 0x61B9411E, 0x16F988FA, 0xF2C65152, 0xA88E5A6D, 0xB019FC65, 0xB9D99EC7, 0x9A1231C3, 0xE70EEAA0, 0xFDB1232B, 0xC7353EB0, 
	0x3069BAD5, 0xCB976D5F, 0x5A0F118F, 0xDC1EEEFD, 0x0A35B689, 0xDE0B7A04, 0x58F4CA9D, 0xE15D5B16, 0x007F3E86, 0x37088980, 0xA507EA32, 0x6FAB9537, 0x17406110, 0x0D8CD6F1, 0xCDAA3B6D, 0xC0BBBE37, 
	0x83613BDA, 0xDB48A363, 0x0B02E931, 0x6FD15CA7, 0x521AFACA, 0x31338431, 0x6ED41A95, 0x6D437890, 0xC39C91F2, 0x9ECCABBD, 0xB5C9A0E6, 0x532FB63C, 0xD2C741C6, 0x07237EA3, 0xA4954B68, 0x4C191D76
};

#define Ch(a, b, c)     (((b^c) & a) ^ c)
#define Maj(x, y, z)    ((x & (y | z)) | (y & z)) //((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

#define xor3b(a,b,c) ((a ^ b) ^ c)

__device__ __forceinline__ uint32_t bsg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x,2),ROTR32(x,13),ROTR32(x,22));
}

__device__ __forceinline__ uint32_t bsg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x,6),ROTR32(x,11),ROTR32(x,25));
}

__device__ __forceinline__ uint32_t ssg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x,7),ROTR32(x,18),(x>>3));
}

__device__ __forceinline__ uint32_t ssg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x,17),ROTR32(x,19),(x>>10));
}

__device__ __forceinline__
static void sha2_step1(const uint32_t a,const uint32_t b,const uint32_t c, uint32_t &d,const uint32_t e,const uint32_t f,const uint32_t g, uint32_t &h,const uint32_t in, const uint32_t Kshared)
{
	const uint32_t t1 = h + bsg2_1(e) + Ch(e, f, g) + Kshared + in;
	h = t1 + bsg2_0(a) + Maj(a, b, c);
	d+= t1;

}

__device__ __forceinline__
static void sha2_step2(const uint32_t a,const uint32_t b,const uint32_t c, uint32_t &d,const uint32_t e,const uint32_t f,const uint32_t g, uint32_t &h, const uint32_t Kshared)
{
	const uint32_t t1 = h + bsg2_1(e) + Ch(e, f, g) + Kshared;
	h = t1 + bsg2_0(a) + Maj(a, b, c);
	d+= t1;

}

__device__ __forceinline__
static void sha256_round_body(uint32_t* in, uint32_t* state,const uint32_t* __restrict__ Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step1(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha2_step1(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha2_step1(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha2_step1(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha2_step1(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha2_step1(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha2_step1(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha2_step1(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha2_step1(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha2_step1(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha2_step1(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha2_step1(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha2_step1(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step1(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step1(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step1(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	#pragma unroll 3
	for (int i=0; i<3; i++)
	{
		#pragma unroll 16
		for (int j = 0; j < 16; j++){
			in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
		}
		sha2_step1(a, b, c, d, e, f, g, h, in[0], Kshared[16 + 16 * i]);
		sha2_step1(h, a, b, c, d, e, f, g, in[1], Kshared[17 + 16 * i]);
		sha2_step1(g, h, a, b, c, d, e, f, in[2], Kshared[18 + 16 * i]);
		sha2_step1(f, g, h, a, b, c, d, e, in[3], Kshared[19 + 16 * i]);
		sha2_step1(e, f, g, h, a, b, c, d, in[4], Kshared[20 + 16 * i]);
		sha2_step1(d, e, f, g, h, a, b, c, in[5], Kshared[21 + 16 * i]);
		sha2_step1(c, d, e, f, g, h, a, b, in[6], Kshared[22 + 16 * i]);
		sha2_step1(b, c, d, e, f, g, h, a, in[7], Kshared[23 + 16 * i]);
		sha2_step1(a, b, c, d, e, f, g, h, in[8], Kshared[24 + 16 * i]);
		sha2_step1(h, a, b, c, d, e, f, g, in[9], Kshared[25 + 16 * i]);
		sha2_step1(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * i]);
		sha2_step1(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * i]);
		sha2_step1(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * i]);
		sha2_step1(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * i]);
		sha2_step1(c, d, e, f, g, h, a, b, in[14], Kshared[30 + 16 * i]);
		sha2_step1(b, c, d, e, f, g, h, a, in[15], Kshared[31 + 16 * i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

__device__ __forceinline__
static void sha256_round_body_final(uint32_t* state,const uint32_t* Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step2(a,b,c,d,e,f,g,h, Kshared[0]);
	sha2_step2(h,a,b,c,d,e,f,g, Kshared[1]);
	sha2_step2(g,h,a,b,c,d,e,f, Kshared[2]);
	sha2_step2(f,g,h,a,b,c,d,e, Kshared[3]);
	sha2_step2(e,f,g,h,a,b,c,d, Kshared[4]);
	sha2_step2(d,e,f,g,h,a,b,c, Kshared[5]);
	sha2_step2(c,d,e,f,g,h,a,b, Kshared[6]);
	sha2_step2(b,c,d,e,f,g,h,a, Kshared[7]);
	sha2_step2(a,b,c,d,e,f,g,h, Kshared[8]);
	sha2_step2(h,a,b,c,d,e,f,g, Kshared[9]);
	sha2_step2(g,h,a,b,c,d,e,f, Kshared[10]);
	sha2_step2(f,g,h,a,b,c,d,e, Kshared[11]);
	sha2_step2(e,f,g,h,a,b,c,d, Kshared[12]);
	sha2_step2(d,e,f,g,h,a,b,c, Kshared[13]);
	sha2_step2(c,d,e,f,g,h,a,b, Kshared[14]);
	sha2_step2(b,c,d,e,f,g,h,a, Kshared[15]);

	#pragma unroll
	for (int i=0; i<2; i++){

		sha2_step2(a, b, c, d, e, f, g, h, Kshared[16 + 16 * i]);
		sha2_step2(h, a, b, c, d, e, f, g, Kshared[17 + 16 * i]);
		sha2_step2(g, h, a, b, c, d, e, f, Kshared[18 + 16 * i]);
		sha2_step2(f, g, h, a, b, c, d, e, Kshared[19 + 16 * i]);
		sha2_step2(e, f, g, h, a, b, c, d, Kshared[20 + 16 * i]);
		sha2_step2(d, e, f, g, h, a, b, c, Kshared[21 + 16 * i]);
		sha2_step2(c, d, e, f, g, h, a, b, Kshared[22 + 16 * i]);
		sha2_step2(b, c, d, e, f, g, h, a, Kshared[23 + 16 * i]);
		sha2_step2(a, b, c, d, e, f, g, h, Kshared[24 + 16 * i]);
		sha2_step2(h, a, b, c, d, e, f, g, Kshared[25 + 16 * i]);
		sha2_step2(g, h, a, b, c, d, e, f, Kshared[26 + 16 * i]);
		sha2_step2(f, g, h, a, b, c, d, e, Kshared[27 + 16 * i]);
		sha2_step2(e, f, g, h, a, b, c, d, Kshared[28 + 16 * i]);
		sha2_step2(d, e, f, g, h, a, b, c, Kshared[29 + 16 * i]);
		sha2_step2(c, d, e, f, g, h, a, b, Kshared[30 + 16 * i]);
		sha2_step2(b, c, d, e, f, g, h, a, Kshared[31 + 16 * i]);
	}
	sha2_step2(a, b, c, d, e, f, g, h, Kshared[16 + 16 * 2]);
	sha2_step2(h, a, b, c, d, e, f, g, Kshared[17 + 16 * 2]);
	sha2_step2(g, h, a, b, c, d, e, f, Kshared[18 + 16 * 2]);
	sha2_step2(f, g, h, a, b, c, d, e, Kshared[19 + 16 * 2]);
	sha2_step2(e, f, g, h, a, b, c, d, Kshared[20 + 16 * 2]);
	sha2_step2(d, e, f, g, h, a, b, c, Kshared[21 + 16 * 2]);
	sha2_step2(c, d, e, f, g, h, a, b, Kshared[22 + 16 * 2]);
	sha2_step2(b, c, d, e, f, g, h, a, Kshared[23 + 16 * 2]);
	sha2_step2(a, b, c, d, e, f, g, h, Kshared[24 + 16 * 2]);
	sha2_step2(h, a, b, c, d, e, f, g, Kshared[25 + 16 * 2]);
	sha2_step2(g, h, a, b, c, d, e, f, Kshared[26 + 16 * 2]);
	sha2_step2(f, g, h, a, b, c, d, e, Kshared[27 + 16 * 2]);
	sha2_step2(e, f, g, h, a, b, c, d, Kshared[28 + 16 * 2]);
	sha2_step2(d, e, f, g, h, a, b, c, Kshared[29 + 16 * 2]);

	state[6]+= g;
	state[7]+= h;
}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(1024,2) /* to force 32 regs */
#else
__launch_bounds__(768,2) /* to force 32 regs */
#endif
void myriadgroestl_gpu_hash_sha(uint32_t threads, uint32_t startNounce, uint32_t* hashBuffer, uint32_t *resNonces,const uint64_t target64){

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t W[16];
		uint32_t *inpHash = &hashBuffer[thread<<4];

		*(uint2x4*)&W[ 0] = __ldg4((uint2x4*)&inpHash[ 0]);
		*(uint2x4*)&W[ 8] = __ldg4((uint2x4*)&inpHash[ 8]);

		uint32_t buf[ 8] = {
			0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
			0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
		};

		sha256_round_body(W,buf,sha256_constantTable);
	
		sha256_round_body_final(buf,sha256_constantTable2);

		#if 0
			// Full sha hash
			#pragma unroll
			for(int k=0; k<8; k++)
				W[k] = cuda_swab32(buf[k]);
		#else
			W[6] = cuda_swab32(buf[6]);
			W[7] = cuda_swab32(buf[7]);
		#endif

		if (*(uint64_t*)&W[6] <= target64){
			uint32_t tmp = atomicExch(&resNonces[0], startNounce + thread);
			if (tmp != UINT32_MAX)
				resNonces[1] = tmp;
		}
	}
}

#define TPB52 512
#define TPB50 512
#define THF 4

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(TPB52, 2)
#else
__launch_bounds__(TPB50, 2)
#endif
void myriadgroestl_gpu_hash_quad(uint32_t threads, uint32_t startNounce, uint32_t *d_hash){

	// durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x)>>2;
	if (thread < threads)
	{
		const uint32_t thr = threadIdx.x & 3;
		// GROESTL
		uint32_t input[8];
		uint32_t other[8];
		uint32_t msgBitsliced[8];
		uint32_t state[8];
		uint32_t output[16];

		*(uint2x4*)input = *(uint2x4*)&c_input[((threadIdx.x & 2)<<3)];
		*(uint2x4*)other = *(uint2x4*)&c_input[(((threadIdx.x+1)&3)<<3)];
		#pragma unroll 8
		for(int k=0; k<8; k++){
//			input[k] = c_input[k+((threadIdx.x & 2)<<3)];
//			other[k] = c_input[k+(((threadIdx.x+1)&3)<<3)];
			other[k] = __shfl(other[k], threadIdx.x & 2, 4);
		}

		if ((thr == 2) || (thr == 3))
			other[4] = cuda_swab32(startNounce + thread);

		uint32_t t;

		const uint32_t perm = (threadIdx.x & 1) ? 0x7362 : 0x5140;

		merge8(msgBitsliced[0], input[0], input[4], perm);
		merge8(msgBitsliced[1], other[0], other[4], perm);
		merge8(msgBitsliced[2], input[1], input[5], perm);
		merge8(msgBitsliced[3], other[1], other[5], perm);
		merge8(msgBitsliced[4], input[2], input[6], perm);
		merge8(msgBitsliced[5], other[2], other[6], perm);
		merge8(msgBitsliced[6], input[3], input[7], perm);
		merge8(msgBitsliced[7], other[3], other[7], perm);

		SWAP1(msgBitsliced[0], msgBitsliced[1]);
		SWAP1(msgBitsliced[2], msgBitsliced[3]);
		SWAP1(msgBitsliced[4], msgBitsliced[5]);
		SWAP1(msgBitsliced[6], msgBitsliced[7]);

		SWAP2(msgBitsliced[0], msgBitsliced[2]);
		SWAP2(msgBitsliced[1], msgBitsliced[3]);
		SWAP2(msgBitsliced[4], msgBitsliced[6]);
		SWAP2(msgBitsliced[5], msgBitsliced[7]);

		SWAP4(msgBitsliced[0], msgBitsliced[4]);
		SWAP4(msgBitsliced[1], msgBitsliced[5]);
		SWAP4(msgBitsliced[2], msgBitsliced[6]);
		SWAP4(msgBitsliced[3], msgBitsliced[7]);

	        groestl512_progressMessage_quad(state, msgBitsliced,thr);

		from_bitslice_quad52(state, output);

		uint2x4* outHash = (uint2x4*)&d_hash[thread<<4];
		
#if __CUDA_ARCH__ <= 500
		output[0] = __byte_perm(output[0], __shfl(output[0], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[2] = __byte_perm(output[2], __shfl(output[2], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[4] = __byte_perm(output[4], __shfl(output[4], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[6] = __byte_perm(output[6], __shfl(output[6], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[8] = __byte_perm(output[8], __shfl(output[8], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[10] = __byte_perm(output[10], __shfl(output[10], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[12] = __byte_perm(output[12], __shfl(output[12], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[14] = __byte_perm(output[14], __shfl(output[14], (threadIdx.x + 1) & 3, 4), 0x2367);
		
		if (thr == 0 || thr == 2){
			output[0 + 1] = __shfl(output[0], (threadIdx.x + 2) & 3, 4);
			output[2 + 1] = __shfl(output[2], (threadIdx.x + 2) & 3, 4);
			output[4 + 1] = __shfl(output[4], (threadIdx.x + 2) & 3, 4);
			output[6 + 1] = __shfl(output[6], (threadIdx.x + 2) & 3, 4);
			output[8 + 1] = __shfl(output[8], (threadIdx.x + 2) & 3, 4);
			output[10 + 1] = __shfl(output[10], (threadIdx.x + 2) & 3, 4);
			output[12 + 1] = __shfl(output[12], (threadIdx.x + 2) & 3, 4);
			output[14 + 1] = __shfl(output[14], (threadIdx.x + 2) & 3, 4);		
			if(thr==0){
				outHash[0] = *(uint2x4*)&output[0];
				outHash[1] = *(uint2x4*)&output[8];
			}
		}
#else
		output[ 0] = __byte_perm(output[0], __shfl(output[0], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[ 1] = __shfl(output[0], (threadIdx.x + 2) & 3, 4);

		output[ 2] = __byte_perm(output[2], __shfl(output[2], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[ 3] = __shfl(output[2], (threadIdx.x + 2) & 3, 4);
		
		output[ 4] = __byte_perm(output[4], __shfl(output[4], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[ 5] = __shfl(output[4], (threadIdx.x + 2) & 3, 4);
		
		output[ 6] = __byte_perm(output[6], __shfl(output[6], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[ 7] = __shfl(output[6], (threadIdx.x + 2) & 3, 4);
		
		output[ 8] = __byte_perm(output[8], __shfl(output[8], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[ 9] = __shfl(output[8], (threadIdx.x + 2) & 3, 4);

		output[10] = __byte_perm(output[10], __shfl(output[10], (threadIdx.x + 1) & 3, 4), 0x0167);
		output[11] = __shfl(output[10], (threadIdx.x + 2) & 3, 4);
		
		output[12] = __byte_perm(output[12], __shfl(output[12], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[13] = __shfl(output[12], (threadIdx.x + 2) & 3, 4);
		
		output[14] = __byte_perm(output[14], __shfl(output[14], (threadIdx.x + 1) & 3, 4), 0x2367);
		output[15] = __shfl(output[14], (threadIdx.x + 2) & 3, 4);

		if(thr==0){
			outHash[0] = *(uint2x4*)&output[0];
			outHash[1] = *(uint2x4*)&output[8];
		}
#endif
    	}
}

// Setup Function
__host__
void myriadgroestl_cpu_init(int thr_id, uint32_t threads)
{
	CUDA_SAFE_CALL(hipMalloc(&d_outputHashes[thr_id], (size_t) 64 * threads));
}

__host__
void myriadgroestl_cpu_free(int thr_id)
{
	hipFree(d_outputHashes[thr_id]);
}

__host__
void myriadgroestl_cpu_setBlock(int thr_id, void *data){

	uint32_t msgBlock[32] = { 0 };
	uint32_t paddedInput[32];
	memcpy(&msgBlock[0], data, 80);
	msgBlock[20] = 0x80;
	msgBlock[31] = 0x01000000;

	for(int thr=0;thr<4;thr++)
		for(int k=0; k<8; k++)
			paddedInput[k+(thr<<3)] = msgBlock[4*k+thr];

	for(int k=0;k<8;k++){
		uint32_t temp = paddedInput[k+(1<<3)];
		paddedInput[k+(1<<3)] = paddedInput[k+(2<<3)];
		paddedInput[k+(2<<3)] = temp;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(c_input), paddedInput, 128);
}

__host__
void myriadgroestl_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_resNounce, const uint64_t target)
{
	// Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
	// mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
	uint32_t tpb = TPB52;
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] <= 500) tpb = TPB50;	
	const dim3 grid((THF*threads + tpb-1)/tpb);
	const dim3 block(tpb);

	myriadgroestl_gpu_hash_quad <<< grid, block >>> (threads, startNounce, d_outputHashes[thr_id]);

	tpb = (device_sm[dev_id] <= 500) ? 768 : 1024;

	dim3 grid2((threads + tpb - 1) / tpb);
	dim3 block2(tpb);
	
	myriadgroestl_gpu_hash_sha <<< grid2, block2 >>> (threads, startNounce, d_outputHashes[thr_id], d_resNounce, target);

}
