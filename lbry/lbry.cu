#include "hip/hip_runtime.h"
/**
 * Lbry CUDA Implementation
 *
 * by tpruvot@github - July 2016
 *
 */

#include <string.h>
#include <stdint.h>

extern "C" {
#include <sph/sph_sha2.h>
#include <sph/sph_ripemd.h>
}

#include <cuda_helper.h>
#include <miner.h>

#define A 64
#define debug_cpu 0

extern "C" void lbry_hash(void* output, const void* input)
{
	uint32_t _ALIGN(A) hashA[16];
	uint32_t _ALIGN(A) hashB[8];
	uint32_t _ALIGN(A) hashC[8];

	sph_sha256_context ctx_sha256;
	sph_sha512_context ctx_sha512;
	sph_ripemd160_context ctx_ripemd;

	sph_sha256_init(&ctx_sha256);
	sph_sha256(&ctx_sha256, input, 112);
	sph_sha256_close(&ctx_sha256, hashA);

	sph_sha256(&ctx_sha256, hashA, 32);
	sph_sha256_close(&ctx_sha256, hashA);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512, hashA, 32);
	sph_sha512_close(&ctx_sha512, hashA);

	sph_ripemd160_init(&ctx_ripemd);
	sph_ripemd160(&ctx_ripemd, hashA, 32);  // sha512 low
	sph_ripemd160_close(&ctx_ripemd, hashB);
	if (debug_cpu) applog_hex(hashB, 20);

	sph_ripemd160(&ctx_ripemd, &hashA[8], 32); // sha512 high
	sph_ripemd160_close(&ctx_ripemd, hashC);
	if (debug_cpu) applog_hex(hashC, 20);

	sph_sha256(&ctx_sha256, hashB, 20);
	sph_sha256(&ctx_sha256, hashC, 20);
	sph_sha256_close(&ctx_sha256, hashA);
	if (debug_cpu) applog_hex(hashA,32);

	sph_sha256(&ctx_sha256, hashA, 32);
	sph_sha256_close(&ctx_sha256, hashA);

	memcpy(output, hashA, 32);
}

/* ############################################################################################################################### */

extern void lbry_sha256_init(int thr_id);
extern void lbry_sha256_free(int thr_id);
extern void lbry_sha256_setBlock_112(uint32_t *pdata);
extern void lbry_sha256d_hash_112(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash);
extern void lbry_sha512_hash_32(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void lbry_sha256d_hash_final(int thr_id, uint32_t threads, uint32_t *d_inputHash, uint32_t *d_resNonce, const uint64_t target64);

extern void lbry_sha256_setBlock_112_merged(uint32_t *pdata);
extern void lbry_merged(int thr_id,uint32_t startNonce, uint32_t threads, uint32_t *d_resNonce, const uint64_t target64);

static __inline uint32_t swab32_if(uint32_t val, bool iftrue) {
	return iftrue ? swab32(val) : val;
}

static bool init[MAX_GPUS] = { 0 };

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];
// nonce position is different
#define LBC_NONCE_OFT32 27

extern "C" int scanhash_lbry(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(A) vhash[8];
	uint32_t _ALIGN(A) endiandata[28];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[LBC_NONCE_OFT32];
	const int swap = 0; // to toggle nonce endian (need kernel change)

	const int dev_id = device_map[thr_id];
	uint32_t default_throughput = (device_sm[dev_id] > 500) ? ((1U<<26)+256*256*256) : ((1U<<23)+256*256*64);
	
	if (strstr(device_name[dev_id], "GTX 1080")) default_throughput=(1<<28);
	else 
	if (strstr(device_name[dev_id], "GTX 1070")) default_throughput=(1<<28);
	else 
	if (strstr(device_name[dev_id], "GTX 1060 6GB")) default_throughput=(1<<28);	
	else 
	if (strstr(device_name[dev_id], "GTX 980")) default_throughput =(1<<28);
	

	uint32_t throughput = cuda_default_throughput(thr_id, default_throughput);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark) {
		ptarget[7] = 0xf;
	}

	if (!init[thr_id]){
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);		

		if(device_sm[dev_id]<=500)
			CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 8 * sizeof(uint64_t) * throughput));

		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)));
		
		CUDA_LOG_ERROR();

		init[thr_id] = true;
	}

	for (int i=0; i < LBC_NONCE_OFT32; i++) {
		be32enc(&endiandata[i], pdata[i]);
	}

	if(device_sm[dev_id]<=500)
		lbry_sha256_setBlock_112(endiandata);
	else
		lbry_sha256_setBlock_112_merged(endiandata);
	hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
	do {
		// Hash with CUDA
		if(device_sm[dev_id]<=500){
			lbry_sha256d_hash_112(thr_id, throughput, pdata[LBC_NONCE_OFT32], d_hash[thr_id]);
			lbry_sha512_hash_32(thr_id, throughput, d_hash[thr_id]);
			lbry_sha256d_hash_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id], *(uint64_t*)&ptarget[6]);
		}else{
			lbry_merged(thr_id,pdata[LBC_NONCE_OFT32], throughput, d_resNonce[thr_id], *(uint64_t*)&ptarget[6]);
		}
		uint32_t resNonces[2] = { UINT32_MAX, UINT32_MAX };
		hipMemcpy(resNonces, d_resNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	
		if (resNonces[ 0] != UINT32_MAX)
		{
			const uint32_t startNonce = pdata[LBC_NONCE_OFT32];
			
			resNonces[ 0]+= startNonce;
			
			endiandata[LBC_NONCE_OFT32] = swab32_if(resNonces[ 0], !swap);
			lbry_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				int res = 1;
				*hashes_done = pdata[LBC_NONCE_OFT32] - first_nonce + throughput;
				work->nonces[0] = swab32_if(resNonces[ 0], swap);
				work_set_target_ratio(work, vhash);
				if (resNonces[ 1] != UINT32_MAX) {
					resNonces[ 1]+=startNonce;
//					if(!opt_quiet)
//						gpulog(LOG_BLUE, thr_id, "Found 2nd nonce %08x", swab32(resNonces[ 1]));
					endiandata[LBC_NONCE_OFT32] = swab32_if(resNonces[ 1], !swap);
					lbry_hash(vhash, endiandata);
					work->nonces[1] = swab32_if(resNonces[ 1], swap);
					if (bn_hash_target_ratio(vhash, ptarget) > work->shareratio[0]) {
						work_set_target_ratio(work, vhash);
						xchg(work->nonces[0], work->nonces[1]);
					}
					res++;
				}
				pdata[LBC_NONCE_OFT32] = work->nonces[0];
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU %08x > %08x!", resNonces[ 0], vhash[7], ptarget[7]);
				hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
			}
		}

		pdata[LBC_NONCE_OFT32] += throughput;

	} while (!work_restart[thr_id].restart && ((uint64_t)throughput + (uint64_t)pdata[LBC_NONCE_OFT32] < (uint64_t)max_nonce));

	*hashes_done = pdata[LBC_NONCE_OFT32] - first_nonce;

	return 0;
}

// cleanup
void free_lbry(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	if(device_sm[device_map[thr_id]]<=500)
		hipFree(d_hash[thr_id]);
		
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
