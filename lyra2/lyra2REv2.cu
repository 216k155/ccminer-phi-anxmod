#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"


static uint2 *d_hash[MAX_GPUS];
static uint2 *d_matrix[MAX_GPUS];

extern void blake256_14round_cpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint2* d_Hash);
extern void blake256_14round_cpu_setBlock_80(const uint32_t *pdata);

extern void keccak256_cpu_hash_32(const int thr_id,const uint32_t threads, uint2* d_hash);
extern void keccak256_cpu_init(int thr_id);
extern void keccak256_cpu_free(int thr_id);

extern void skein256_cpu_hash_32(const uint32_t threads, uint2 *d_hash);
extern void skein256_cpu_init(int thr_id);

extern void cubehash256_cpu_hash_32(const uint32_t threads, uint2* d_hash);

extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads,uint2* DMatrix, uint2 *d_Hash);
extern void lyra2v2_cpu_init(int thr_id, uint2 *hash);

extern void bmw256_setTarget(const void *ptarget);
extern void bmw256_cpu_init(int thr_id);
extern void bmw256_cpu_free(int thr_id);
extern void bmw_set_output(int thr_id);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads,uint2 *g_hash, uint32_t *resultnonces, const uint2 target);

void lyra2v2_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);

	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 4, 4);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);

	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("lyra2 %s %08x %08x %08x %08x...%08x... \n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2v2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500) ? 22 : 20;
	uint32_t throughput = cuda_default_throughput(dev_id, 1UL << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);			
			CUDA_LOG_ERROR();
		}

		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		
		cuda_get_arch(dev_id);
		skein256_cpu_init(thr_id);
//		keccak256_cpu_init(thr_id);
		bmw256_cpu_init(thr_id);

		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], 4 * 4 * sizeof(uint2) * throughput));
//		lyra2v2_cpu_init(thr_id, d_matrix[thr_id]);
		
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * throughput));
//		api_set_throughput(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_14round_cpu_setBlock_80(pdata);
	bmw_set_output(thr_id);
	do {
		uint32_t foundNonces[2] = { 0, 0 };

		blake256_14round_cpu_hash_80(throughput, pdata[19], d_hash[thr_id]);
		keccak256_cpu_hash_32(thr_id,throughput, d_hash[thr_id]);
		cubehash256_cpu_hash_32(throughput, d_hash[thr_id]);
		lyra2v2_cpu_hash_32(thr_id, throughput,d_matrix[thr_id],d_hash[thr_id]);
		skein256_cpu_hash_32(throughput, d_hash[thr_id]);
		cubehash256_cpu_hash_32(throughput, d_hash[thr_id]);
		bmw256_cpu_hash_32(thr_id, throughput, d_hash[thr_id], foundNonces, *(uint2*)&ptarget[6]);

		if (foundNonces[0] != 0)
		{
			const uint32_t startNounce = pdata[19];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], startNounce + foundNonces[0]);
			lyra2v2_hash(vhash64, endiandata);
			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				work_set_target_ratio(work, vhash64);
				pdata[19] = startNounce + foundNonces[0];
				// check if there was another one...
				if (foundNonces[1] != 0)
				{
					be32enc(&endiandata[19], (pdata[19]-foundNonces[ 0])+foundNonces[1]);
					lyra2v2_hash(vhash64, endiandata);
					pdata[21] = startNounce + foundNonces[1];
//					if(!opt_quiet)
//						applog(LOG_BLUE,"Found 2nd nonce: %08X",pdata[21]);
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0]) {
						work_set_target_ratio(work, vhash64);
						xchg(pdata[19], pdata[21]);
					}
					res=2;
				}
				return res;
			}
			else
			{
				if(vhash64[7]>ptarget[ 7])
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonces[0]);
				bmw_set_output(thr_id);
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && !abort_flag);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2v2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	bmw256_cpu_free(thr_id);
//	keccak256_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
