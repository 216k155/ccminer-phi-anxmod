#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint2* d_hash[MAX_GPUS];
static uint2* d_matrix[MAX_GPUS];

extern void blake256_14round_cpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint2* d_Hash);
extern void blake256_14round_cpu_setBlock_80(const uint32_t *pdata);

extern void keccak256_cpu_hash_32(const int thr_id,const uint32_t threads, uint2* d_hash);
extern void keccak256_cpu_init(int thr_id);
extern void keccak256_cpu_free(int thr_id);

extern void skein256_cpu_init(int thr_id);
extern void skein256_cpu_hash_32(const uint32_t threads, uint2 *d_hash);

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint2* d_matrix);
extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint2* d_outputHash);

extern void groestl256_cpu_init(int thr_id, uint32_t threads);
extern void groestl256_cpu_free(int thr_id);
extern void groestl256_setTarget(const void *ptarget);
extern void groestl256_set_output(int thr_id);
extern void groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint2 *d_Hash, uint32_t *resultnonces);


#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("lyra %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

extern "C" void lyra2re_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{

	const int dev_id = device_map[thr_id];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[dev_id] > 500 ) ? 19 : 17;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		
		keccak256_cpu_init(thr_id);
		skein256_cpu_init(thr_id);
		groestl256_cpu_init(thr_id, throughput);

		// DMatrix
//		size_t matrix_sz = sizeof(uint64_t) * 4 * 4;
//		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
//		lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : 16 * 8 * 8 * sizeof(uint2);
		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
		lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);
//		hipMalloc(&d_matrix[thr_id], (size_t)16 * 8 * 8 * sizeof(uint2) * throughput);
//		lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t _ALIGN(128) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_14round_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);
	groestl256_set_output(thr_id);
	do {
		uint32_t foundNonces[2];

		blake256_14round_cpu_hash_80(throughput, pdata[19], d_hash[thr_id]);
		keccak256_cpu_hash_32(thr_id,throughput, d_hash[thr_id]);
		lyra2_cpu_hash_32(thr_id, throughput, d_hash[thr_id]);
		skein256_cpu_hash_32(throughput, d_hash[thr_id]);
		groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonces);

		if (foundNonces[ 0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];

			be32enc(&endiandata[19], foundNonces[ 0]);
			lyra2re_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;

				work_set_target_ratio(work, vhash64);
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = foundNonces[ 0];
				if (foundNonces[ 1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], foundNonces[ 1]);
					lyra2re_hash(vhash64, endiandata);
//					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
//						applog(LOG_NOTICE,"Legit extranonce");
						if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0])
							work_set_target_ratio(work, vhash64);
						pdata[21] = foundNonces[ 1];
						res++;
//					}
//					if(!opt_quiet)
//						applog(LOG_BLUE,"Found 2nd nonce: %08X",pdata[21]);
				}
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonces[ 0]);
				groestl256_set_output(thr_id);
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	keccak256_cpu_free(thr_id);
	groestl256_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
