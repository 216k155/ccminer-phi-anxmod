#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"

#include "miner.h"
}

#include "cuda_helper.h"

extern uint32_t *h_nounce[MAX_GPUS];

extern void keccak256_cpu_init(int thr_id);
extern void keccak256_cpu_free(int thr_id);
extern void keccak256_setBlock_80(uint64_t *PaddedMessage80);
extern void keccak256_setOutput(int thr_id);
extern void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce,const uint2 highTarget);

// CPU Hash
extern "C" void keccak256_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_keccak_context ctx_keccak;

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_keccak256(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done){

	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] >= 520) ? 28 : 25;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

//	const uint64_t highTarget = *(uint64_t*)&ptarget[6];
	const uint2 highTarget = make_uint2(ptarget[6],ptarget[7]);
	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		keccak256_cpu_init(thr_id);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	keccak256_setBlock_80((uint64_t*)endiandata);
	keccak256_setOutput(thr_id);
	int rc = 0;
	do {
		keccak256_cpu_hash_80(thr_id, throughput, pdata[19],highTarget);
		if (h_nounce[thr_id][0] != UINT32_MAX && bench_algo < 0)
		{
			uint32_t _ALIGN(64) vhash64[8];
			be32enc(&endiandata[19], h_nounce[thr_id][0]);
			keccak256_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				*hashes_done = pdata[19] - first_nonce + throughput;
				work_set_target_ratio(work, vhash64);
				pdata[19] = h_nounce[thr_id][0];
				rc = 1;
				if(h_nounce[thr_id][1]!=UINT32_MAX){
//					if(!opt_quiet)
//						gpulog(LOG_BLUE,dev_id,"Found 2nd nonce %08x !", h_nounce[thr_id][1]);
					be32enc(&endiandata[19], h_nounce[thr_id][1]);
					keccak256_hash(vhash64, endiandata);
					pdata[21] = h_nounce[thr_id][1];
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0]) {
						work_set_target_ratio(work, vhash64);
						xchg(pdata[19], pdata[21]);
					}
					rc = 2;
				}
				return rc;
			}
			else{
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_nounce[thr_id][0]);
				keccak256_setOutput(thr_id);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;

	return rc;

}

// cleanup
extern "C" void free_keccak256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	keccak256_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
